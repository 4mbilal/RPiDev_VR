#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include "MWCNNLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#ifdef RANDOM
#include <hiprand.h>
 hiprandGenerator_t TYgANfbwgYWWZKKtdxCC; void 
curand_call_line_file(hiprandStatus_t tYWUxNVtgBrSjkBemGfF, const int 
aJTwGElOoWpBrmCfheqQ, const char* RtogJCavwOREhELwknZy) { if (tYWUxNVtgBrSjkBemGfF != 
HIPRAND_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"%d at line: %d, file: %s\n", tYWUxNVtgBrSjkBemGfF, aJTwGElOoWpBrmCfheqQ, 
RtogJCavwOREhELwknZy); throw std::runtime_error(buffer); } }
#endif
 void call_cuda_free(float* mem, const int aJTwGElOoWpBrmCfheqQ, const char* 
RtogJCavwOREhELwknZy) { if (!mem) { return; } hipError_t tYWUxNVtgBrSjkBemGfF = 
hipFree(mem); } float* malloc_call_line_file(size_t msize, const int 
aJTwGElOoWpBrmCfheqQ, const char *RtogJCavwOREhELwknZy) { float * mem = 
(float*)malloc(msize); if (!mem) { char buffer[100]; int numElem = 
sprintf(buffer, "%s at line: %d, file: %s\n", "Memory allocation failed. ", 
aJTwGElOoWpBrmCfheqQ, RtogJCavwOREhELwknZy); throw std::runtime_error(buffer); } return 
mem; } void cuda_call_line_file(hipError_t tYWUxNVtgBrSjkBemGfF, const int 
aJTwGElOoWpBrmCfheqQ, const char* RtogJCavwOREhELwknZy) { if (tYWUxNVtgBrSjkBemGfF != 
hipSuccess) { char buffer[100]; int numElem = sprintf(buffer, 
"Cuda Error %d(%s) at line: %d, file: %s\n", tYWUxNVtgBrSjkBemGfF, 
hipGetErrorString(tYWUxNVtgBrSjkBemGfF), aJTwGElOoWpBrmCfheqQ, RtogJCavwOREhELwknZy); 
tYWUxNVtgBrSjkBemGfF = hipGetLastError();  throw std::runtime_error(buffer); } } 
void cudnn_call_line_file(hipdnnStatus_t tYWUxNVtgBrSjkBemGfF, const int 
aJTwGElOoWpBrmCfheqQ, const char* RtogJCavwOREhELwknZy) { if (tYWUxNVtgBrSjkBemGfF != 
HIPDNN_STATUS_SUCCESS) {  char buffer[100]; int numElem = sprintf(buffer, 
"CuDNN Error %d(%s) at line: %d, file: %s\n", tYWUxNVtgBrSjkBemGfF, 
hipdnnGetErrorString(tYWUxNVtgBrSjkBemGfF), aJTwGElOoWpBrmCfheqQ, RtogJCavwOREhELwknZy); 
throw std::runtime_error(buffer); } } 
MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : aFIIJSbJDJUndqPzwinJ(layer) , dwFpvfypaTkJiYAULzFs(ntwk_impl) , 
RgALmBtPIZWDevjZBUHy(0.0) , RFQXHGHdWUKqrdBFLaiy(1.0) , QKDZxzoIvpFPYOFxhkKX(-1.0) , 
FCVkOsYRLlakBfSVryaq(0) { } MWCNNLayerImpl::~MWCNNLayerImpl() { 
for(std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
lqYqXvaqiZxCnTfQwVBT.begin(); it != lqYqXvaqiZxCnTfQwVBT.end(); ++it) { 
delete it->second; it->second = 0; } } ITensor* 
MWCNNLayerImpl::getInputITensor(int inputIdx) { MWTensor* ipTensor = 
getLayer()->getInputTensor(inputIdx); assert(ipTensor); return 
getITensor(ipTensor); } ITensor* MWCNNLayerImpl::getITensor(MWTensor* tensor) { 
if (tensor->getOwner()->getImpl() == NULL) { return 
getITensor(tensor->getOwner()->getInputTensor(0)); } else { return 
tensor->getOwner()->getImpl()->getOpTensorPtr(tensor->getSourcePortIndex()); } 
} hipdnnTensorDescriptor_t* MWCNNLayerImpl::getOutputDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
lqYqXvaqiZxCnTfQwVBT.find(index); if (it == lqYqXvaqiZxCnTfQwVBT.end()) { 
hipdnnTensorDescriptor_t* tmp = new hipdnnTensorDescriptor_t; 
lqYqXvaqiZxCnTfQwVBT[index] = tmp; assert(tmp != 0); return tmp; } else { 
assert(it->second != 0); return it->second; } } void 
MWCNNLayerImpl::deallocateOutputData(){ for (int i = 0; i < 
getLayer()->getNumOutputs(); ++i){ MWTensor* opTensor = 
getLayer()->getOutputTensor(i); float* data = opTensor->getData<float>(); if 
(data) { CUDA_FREE_CALL(data); opTensor->setData((float*)NULL); } } } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getCuDNNDescriptor(MWTensor* tensor) { 
return tensor->getOwner()->getImpl()->getOutputDescriptor( 
tensor->getSourcePortIndex()); } int MWCNNLayerImpl::pluginEnqueueImpl(const 
void* const * , void** ){ assert(false); return 0; } 
MWPluginInterfaceImpl::MWPluginInterfaceImpl(MWCNNLayerImpl* 
PsSZzscVKwYLIATdyqkh) : m_cnnLayerImpl(PsSZzscVKwYLIATdyqkh){} Dims 
MWPluginInterfaceImpl::getOutputDimensions(int index, const Dims* , int ) { if 
(!m_cnnLayerImpl->dwFpvfypaTkJiYAULzFs->isSequenceNetwork){ int 
PiMNTwjpqwsGWomVWqdO = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getChannels(); int 
UGusHMMXcwaKODbaZkQs = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getHeight(); int 
znJVDnWdGXAXoBVlQhwT = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getWidth(); return 
DimsCHW(PiMNTwjpqwsGWomVWqdO, UGusHMMXcwaKODbaZkQs, znJVDnWdGXAXoBVlQhwT); }
#if (NV_TENSORRT_MAJOR >= 5)
 else{ int sPCEmfHYfjaRzyVvCKeA = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getSequenceLength(); int 
PiMNTwjpqwsGWomVWqdO = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getChannels(); int 
NhdIzzqqVxMjekDIWciw = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getBatchSize(); return 
Dims3(sPCEmfHYfjaRzyVvCKeA, NhdIzzqqVxMjekDIWciw, PiMNTwjpqwsGWomVWqdO); }
#endif 
 } void MWPluginInterfaceImpl::configure(const Dims* inputDims, int nbInputs, 
const Dims* outputDims, int nbOutputs, int ) { assert(inputDims->nbDims == 3);  
assert(outputDims->nbDims == 3);  assert(nbInputs == 
m_cnnLayerImpl->getLayer()->getNumInputs()); assert(nbOutputs == 
m_cnnLayerImpl->getLayer()->getNumOutputs()); } int 
MWPluginInterfaceImpl::getNbOutputs() const{ return 
m_cnnLayerImpl->getLayer()->getNumOutputs(); } int 
MWPluginInterfaceImpl::enqueue(int , const void* const* inputs, void** outputs, 
void* , hipStream_t ) { m_cnnLayerImpl->pluginEnqueueImpl(inputs,outputs); 
return 0; } MWInputLayerImpl::MWInputLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int crKSAZwnyiinNFYODxoN, int ThkGOmtrxiMfUeOSxFsN, int 
wDZXrivCbTUzkLKxiPUh, int PIyXElJqMZoWKemWyTOa, int , const char* , int ) : 
MWCNNLayerImpl(layer, ntwk_impl) { MWTensor* opTensor = 
getLayer()->getOutputTensor(0); float * QHUGvHzeHXyFElIiOliL; 
CUDA_CALL(hipMalloc((void**)&QHUGvHzeHXyFElIiOliL, sizeof(float) * ThkGOmtrxiMfUeOSxFsN * 
wDZXrivCbTUzkLKxiPUh * PIyXElJqMZoWKemWyTOa * crKSAZwnyiinNFYODxoN)); InputLayerITensor = 
dwFpvfypaTkJiYAULzFs->network->addInput( "data", DataType::kFLOAT, 
DimsCHW{PIyXElJqMZoWKemWyTOa, ThkGOmtrxiMfUeOSxFsN, wDZXrivCbTUzkLKxiPUh}); 
setOpTensorPtr(InputLayerITensor); opTensor->setData(QHUGvHzeHXyFElIiOliL); } void 
MWInputLayerImpl::cleanup() { for (int idx = 0; idx < 
aFIIJSbJDJUndqPzwinJ->getNumOutputs(); idx++) { float* data = 
aFIIJSbJDJUndqPzwinJ->getOutputTensor(idx)->getData<float>(); if (data) { 
CUDA_FREE_CALL(data); } } } MWReLULayerImpl::MWReLULayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int , int ) : MWCNNLayerImpl(layer, ntwk_impl) 
, iReLULayer(0) { ITensor* prevLayerTensor = getInputITensor(0); iReLULayer = 
dwFpvfypaTkJiYAULzFs->network->addActivation(*prevLayerTensor, 
ActivationType::kRELU); iReLULayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iReLULayer->getOutput(0)); } 
MWNormLayerImpl::MWNormLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, unsigned INKFbkrHldYkZFmALnfC,  double ATYqlAsSnRELrakAbCoK,  
double AjhVZuQXURJimwbnYqDF,  double EMtxAWxHxCcPIkaNDIHM, int ) : MWCNNLayerImpl(layer, 
ntwk_impl) { ITensor* prevLayerTensor = getInputITensor(0); iNormLayer = 
dwFpvfypaTkJiYAULzFs->network->addLRN(*prevLayerTensor, 
INKFbkrHldYkZFmALnfC, ATYqlAsSnRELrakAbCoK, AjhVZuQXURJimwbnYqDF, EMtxAWxHxCcPIkaNDIHM); 
iNormLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iNormLayer->getOutput(0)); } void __global__ 
__launch_bounds__(1024) MWSetDyForBackPropImpl(float * RKrEonnJBdcnwoJXOHNM, const int 
gJJWRjXklapoEujuiRhJ) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
gJJWRjXklapoEujuiRhJ; i+= blockDim.x*gridDim.x) { RKrEonnJBdcnwoJXOHNM[i] = i+1; } } 
void __global__ __launch_bounds__(1024) doMWMaxPoolingLayerImpl(float * 
UdXKaaSyRlPmZWCVHODj, float * UROOthsHWeMcNycRifoq, const int 
EWUFPRDanwwTdrjmLomh) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
EWUFPRDanwwTdrjmLomh; i+= blockDim.x*gridDim.x) { if 
(static_cast<int>(UdXKaaSyRlPmZWCVHODj[i]) != 0){ 
UROOthsHWeMcNycRifoq[static_cast<int>(UdXKaaSyRlPmZWCVHODj[i])-1] = 
i; } } } MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int HUlJqbQweWuEmHtzurnN,  int HXjVDkKHrBbbPfJOPKXp,  
int IGBjAMvMJXqrubGDtvyq,  int ILoEtPdZOaUSKUyteZTa, int GtFSKuDmLreppbjSISoU, int 
GbHRuweETkejIMGyqHDI,  int GmRRxuYauzGdhIlgciAT, int GrowsTaKrpHVUZdgZeJW, bool 
IggeMCRfncIoKgBcfyKF, int MW_mangled_, const std::vector<int>& ) : 
MWCNNLayerImpl(layer, ntwk_impl) , iMaxPoolingLayer(0) , 
HKStLBswJlYYprZPPGQx(0) , mJHKRLbEVeCWkBgMqjYL(0) , 
TwiaHttwApyaipMEKPSg(IggeMCRfncIoKgBcfyKF) { ITensor* prevLayerTensor = 
getInputITensor(0); if (!TwiaHttwApyaipMEKPSg && (GtFSKuDmLreppbjSISoU == 
GbHRuweETkejIMGyqHDI) && (GmRRxuYauzGdhIlgciAT == GrowsTaKrpHVUZdgZeJW)){ 
iMaxPoolingLayer = dwFpvfypaTkJiYAULzFs->network->addPooling( *prevLayerTensor, 
PoolingType::kMAX, DimsHW{HUlJqbQweWuEmHtzurnN, HXjVDkKHrBbbPfJOPKXp}); 
iMaxPoolingLayer->setStride(DimsHW{IGBjAMvMJXqrubGDtvyq, ILoEtPdZOaUSKUyteZTa}); 
iMaxPoolingLayer->setPadding(DimsHW{GtFSKuDmLreppbjSISoU, 
GmRRxuYauzGdhIlgciAT}); 
iMaxPoolingLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iMaxPoolingLayer->getOutput(0)); } else{ 
pluginSetup(HUlJqbQweWuEmHtzurnN, HXjVDkKHrBbbPfJOPKXp, IGBjAMvMJXqrubGDtvyq, 
ILoEtPdZOaUSKUyteZTa, GtFSKuDmLreppbjSISoU, GmRRxuYauzGdhIlgciAT); 
mJHKRLbEVeCWkBgMqjYL = new MWPluginInterfaceImpl(this); HKStLBswJlYYprZPPGQx = 
dwFpvfypaTkJiYAULzFs->network->addPlugin(&prevLayerTensor, 1, 
*mJHKRLbEVeCWkBgMqjYL); setOpTensorPtr(HKStLBswJlYYprZPPGQx->getOutput(0),0); 
HKStLBswJlYYprZPPGQx->setName(getLayer()->getName().c_str()); if 
(TwiaHttwApyaipMEKPSg) setOpTensorPtr(HKStLBswJlYYprZPPGQx->getOutput(1),1); 
} } float* MWMaxPoolingLayerImpl::getIndexData() { return NULL; } void 
MWMaxPoolingLayerImpl::cleanup() { if (mJHKRLbEVeCWkBgMqjYL){ 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(mMiQeYHoTbNUGLyNakei)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*XOJRvKzQwSaZobhyUoOi)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*jNxFsuLXTFYGOUlfRwLW)); } if 
(TwiaHttwApyaipMEKPSg) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*jodrcfdVqEtXMUFJulgi)); 
CUDA_FREE_CALL(UdXKaaSyRlPmZWCVHODj); CUDA_FREE_CALL(RKrEonnJBdcnwoJXOHNM); } 
} void MWMaxPoolingLayerImpl::pluginSetup(int HUlJqbQweWuEmHtzurnN, int 
HXjVDkKHrBbbPfJOPKXp, int IGBjAMvMJXqrubGDtvyq, int ILoEtPdZOaUSKUyteZTa, int 
GtFSKuDmLreppbjSISoU, int GmRRxuYauzGdhIlgciAT){ MWTensor* ipTensor = 
getLayer()->getInputTensor();  
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&mMiQeYHoTbNUGLyNakei)); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(mMiQeYHoTbNUGLyNakei, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, HUlJqbQweWuEmHtzurnN, HXjVDkKHrBbbPfJOPKXp, 
GtFSKuDmLreppbjSISoU, GmRRxuYauzGdhIlgciAT, IGBjAMvMJXqrubGDtvyq, 
ILoEtPdZOaUSKUyteZTa)); XOJRvKzQwSaZobhyUoOi = new hipdnnTensorDescriptor_t; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(XOJRvKzQwSaZobhyUoOi)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*XOJRvKzQwSaZobhyUoOi, HIPDNN_TENSOR_NCHW,  
HIPDNN_DATA_FLOAT, ipTensor->getBatchSize(),  ipTensor->getChannels(),  
ipTensor->getHeight(),  ipTensor->getWidth()));  int crKSAZwnyiinNFYODxoN, 
PIyXElJqMZoWKemWyTOa, ThkGOmtrxiMfUeOSxFsN, wDZXrivCbTUzkLKxiPUh; 
CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(mMiQeYHoTbNUGLyNakei, 
*XOJRvKzQwSaZobhyUoOi, &crKSAZwnyiinNFYODxoN ,&PIyXElJqMZoWKemWyTOa, &ThkGOmtrxiMfUeOSxFsN, 
&wDZXrivCbTUzkLKxiPUh)); ThkGOmtrxiMfUeOSxFsN = getLayer()->getOutputTensor(0)->getHeight(); 
wDZXrivCbTUzkLKxiPUh = getLayer()->getOutputTensor(0)->getWidth(); jNxFsuLXTFYGOUlfRwLW = 
new hipdnnTensorDescriptor_t; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(jNxFsuLXTFYGOUlfRwLW)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*jNxFsuLXTFYGOUlfRwLW, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, crKSAZwnyiinNFYODxoN, PIyXElJqMZoWKemWyTOa, ThkGOmtrxiMfUeOSxFsN, wDZXrivCbTUzkLKxiPUh)); if 
(TwiaHttwApyaipMEKPSg){ jodrcfdVqEtXMUFJulgi = new hipdnnTensorDescriptor_t; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(jodrcfdVqEtXMUFJulgi)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*jodrcfdVqEtXMUFJulgi, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, crKSAZwnyiinNFYODxoN, PIyXElJqMZoWKemWyTOa, ThkGOmtrxiMfUeOSxFsN, wDZXrivCbTUzkLKxiPUh)); 
assert((PIyXElJqMZoWKemWyTOa == ipTensor->getChannels()) && (crKSAZwnyiinNFYODxoN == 
ipTensor->getBatchSize()));  ekFKUFSJOBakwDuHENjA = 
(ipTensor->getHeight())*(ipTensor->getWidth())*(ipTensor->getChannels())*(ipTensor->getBatchSize()); 
CUDA_CALL(hipMalloc((void**)&UdXKaaSyRlPmZWCVHODj, 
sizeof(float)*ekFKUFSJOBakwDuHENjA)); gJJWRjXklapoEujuiRhJ = 
wDZXrivCbTUzkLKxiPUh*ThkGOmtrxiMfUeOSxFsN*PIyXElJqMZoWKemWyTOa*crKSAZwnyiinNFYODxoN; 
CUDA_CALL(hipMalloc((void**)&RKrEonnJBdcnwoJXOHNM, 
sizeof(float)*gJJWRjXklapoEujuiRhJ)); int vVyVzWKKaCvGClCSagOb = 
(gJJWRjXklapoEujuiRhJ < 1024) ? gJJWRjXklapoEujuiRhJ : 1024; int 
OuTwywxKeMgznElXdjGp = (gJJWRjXklapoEujuiRhJ + vVyVzWKKaCvGClCSagOb - 
1)/vVyVzWKKaCvGClCSagOb; 
MWSetDyForBackPropImpl<<<OuTwywxKeMgznElXdjGp, 
vVyVzWKKaCvGClCSagOb>>>( RKrEonnJBdcnwoJXOHNM, gJJWRjXklapoEujuiRhJ); } } int 
MWMaxPoolingLayerImpl::pluginEnqueueImpl(const void* const * inputs, void** 
outputs){ 
CUDNN_CALL(hipdnnPoolingForward(*dwFpvfypaTkJiYAULzFs->getCudnnHandle(), 
mMiQeYHoTbNUGLyNakei, getOnePtr(), *XOJRvKzQwSaZobhyUoOi, (float*)inputs[0], 
getZeroPtr(), *jNxFsuLXTFYGOUlfRwLW, (float*)outputs[0])); if 
(TwiaHttwApyaipMEKPSg) { MWTensor* ipTensor = getLayer()->getInputTensor(); 
CUDNN_CALL(hipdnnPoolingBackward(*dwFpvfypaTkJiYAULzFs->getCudnnHandle(), 
mMiQeYHoTbNUGLyNakei, getOnePtr(), *jNxFsuLXTFYGOUlfRwLW, (float*)outputs[0], 
*jNxFsuLXTFYGOUlfRwLW, RKrEonnJBdcnwoJXOHNM, *XOJRvKzQwSaZobhyUoOi, (float*)inputs[0], 
getZeroPtr(), *XOJRvKzQwSaZobhyUoOi, UdXKaaSyRlPmZWCVHODj)); int 
vVyVzWKKaCvGClCSagOb = (ekFKUFSJOBakwDuHENjA < 1024) ? ekFKUFSJOBakwDuHENjA : 
1024; int OuTwywxKeMgznElXdjGp = (ekFKUFSJOBakwDuHENjA + 
vVyVzWKKaCvGClCSagOb - 1)/vVyVzWKKaCvGClCSagOb; 
doMWMaxPoolingLayerImpl<<<OuTwywxKeMgznElXdjGp, 
vVyVzWKKaCvGClCSagOb>>>( UdXKaaSyRlPmZWCVHODj, 
(float*)outputs[1], ekFKUFSJOBakwDuHENjA); } return 0; } 
MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* ntwk_impl, 
int EpwuhXsRcwdqXSjBpUeO, const char* zdjrTcsHRcPcpoVFAMfT,  const 
char* OWgntZrUmlZXHAsNObcq, int ) : MWCNNLayerImpl(layer, ntwk_impl) , 
iFCLayer(0) { MWTensor* opTensor = getLayer()->getOutputTensor(0); MWTensor* 
ipTensor = getLayer()->getInputTensor(0); wDZXrivCbTUzkLKxiPUh = 
(float*)calloc(EpwuhXsRcwdqXSjBpUeO * opTensor->getChannels(), 
sizeof(float)); NSzdekOvRhMhRCXdWsdY = (float*)calloc(opTensor->getChannels(), 
sizeof(float)); int eWYFXrUazhqiEIscccda = EpwuhXsRcwdqXSjBpUeO * 
opTensor->getChannels();  loadWeights(eWYFXrUazhqiEIscccda, zdjrTcsHRcPcpoVFAMfT); 
loadBias(OWgntZrUmlZXHAsNObcq); ITensor* prevLayerITensor = getInputITensor(0); 
filt_weights.values = wDZXrivCbTUzkLKxiPUh; filt_weights.count = 
EpwuhXsRcwdqXSjBpUeO * opTensor->getChannels(); filt_weights.type = 
DataType::kFLOAT; filt_bias.values = NSzdekOvRhMhRCXdWsdY; filt_bias.count = 
opTensor->getChannels(); filt_bias.type = DataType::kFLOAT; if 
(!dwFpvfypaTkJiYAULzFs->isSequenceNetwork){ iFCLayer = 
dwFpvfypaTkJiYAULzFs->network->addFullyConnected( *prevLayerITensor, 
opTensor->getChannels(), filt_weights, filt_bias); 
iFCLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iFCLayer->getOutput(0)); }
#if (NV_TENSORRT_MAJOR >= 5)
 else{ auto shuffleLayer = 
dwFpvfypaTkJiYAULzFs->network->addShuffle(*prevLayerITensor); 
assert(shuffleLayer); shuffleLayer->setFirstTranspose(Permutation{1, 0, 2}); 
auto fcwts = dwFpvfypaTkJiYAULzFs->network->addConstant(Dims3(1, 
opTensor->getChannels(), EpwuhXsRcwdqXSjBpUeO), filt_weights);
#if (NV_TENSORRT_MAJOR >= 5 && NV_TENSORRT_MINOR >= 1)
 auto matrixMultLayer = dwFpvfypaTkJiYAULzFs->network->addMatrixMultiply( 
*fcwts->getOutput(0), MatrixOperation::kNONE, *shuffleLayer->getOutput(0), MatrixOperation::kTRANSPOSE);
#else
 auto matrixMultLayer = dwFpvfypaTkJiYAULzFs->network->addMatrixMultiply( 
*fcwts->getOutput(0), false, *shuffleLayer->getOutput(0), true);
#endif
 assert(matrixMultLayer != nullptr); auto fcbias = 
dwFpvfypaTkJiYAULzFs->network->addConstant(Dims3(1, opTensor->getChannels(), 1), 
filt_bias); auto elementWiseLayer = 
dwFpvfypaTkJiYAULzFs->network->addElementWise(*matrixMultLayer->getOutput(0), 
*fcbias->getOutput(0), ElementWiseOperation::kSUM); assert(elementWiseLayer != 
nullptr); shuffleLayer = 
dwFpvfypaTkJiYAULzFs->network->addShuffle(*elementWiseLayer->getOutput(0)); 
assert(shuffleLayer); shuffleLayer->setFirstTranspose(Permutation{2, 0, 1}); 
setOpTensorPtr(shuffleLayer->getOutput(0)); }
#endif
 } void MWFCLayerImpl::loadWeights(int eWYFXrUazhqiEIscccda, const char* 
RuGYRQXjIMQJrbgoRUxZ) { MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* ipTensor = 
fcLayer->getInputTensor(0); FILE* SmibqCQPbtzycGEpwhpN = 
MWCNNLayer::openBinaryFile(RuGYRQXjIMQJrbgoRUxZ); assert(SmibqCQPbtzycGEpwhpN); 
call_fread(wDZXrivCbTUzkLKxiPUh, sizeof(float), eWYFXrUazhqiEIscccda, SmibqCQPbtzycGEpwhpN, 
RuGYRQXjIMQJrbgoRUxZ); if (ipTensor->getHeight() != 1 && ipTensor->getWidth() != 
1) { float* PAwKCndEJEByqwNZnPgb = (float*)malloc(sizeof(float) * 
ipTensor->getHeight() * ipTensor->getWidth()); for (int k = 0; k < 
eWYFXrUazhqiEIscccda / ipTensor->getHeight() / ipTensor->getWidth(); k++) { for (int 
i = 0; i < ipTensor->getHeight() * ipTensor->getWidth(); i++) { 
PAwKCndEJEByqwNZnPgb[i] = wDZXrivCbTUzkLKxiPUh[k * ipTensor->getHeight() * 
ipTensor->getWidth() + i]; } for (int j = 0; j < ipTensor->getHeight(); j++) 
for (int i = 0; i < ipTensor->getWidth(); i++) { wDZXrivCbTUzkLKxiPUh[k * 
ipTensor->getHeight() * ipTensor->getWidth() + j * ipTensor->getWidth() + i] = 
PAwKCndEJEByqwNZnPgb[j + i * ipTensor->getHeight()]; } } 
free(PAwKCndEJEByqwNZnPgb); } fclose(SmibqCQPbtzycGEpwhpN); } void 
MWFCLayerImpl::loadBias(const char* RuGYRQXjIMQJrbgoRUxZ) { MWTensor* opTensor = 
getLayer()->getOutputTensor(0); FILE* SmibqCQPbtzycGEpwhpN = 
MWCNNLayer::openBinaryFile(RuGYRQXjIMQJrbgoRUxZ); assert(SmibqCQPbtzycGEpwhpN); int 
eWYFXrUazhqiEIscccda = opTensor->getChannels();  call_fread(NSzdekOvRhMhRCXdWsdY, 
sizeof(float), eWYFXrUazhqiEIscccda, SmibqCQPbtzycGEpwhpN, RuGYRQXjIMQJrbgoRUxZ); 
fclose(SmibqCQPbtzycGEpwhpN); } void MWFCLayerImpl::cleanup() { free(wDZXrivCbTUzkLKxiPUh); 
free(NSzdekOvRhMhRCXdWsdY); } MWSoftmaxLayerImpl::MWSoftmaxLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int ) : MWCNNLayerImpl(layer, ntwk_impl) , 
iSoftmaxLayer(0) { MWTensor* opTensor = getLayer()->getOutputTensor(0); 
ITensor* prevLayerTensor = getInputITensor(0); if 
(!dwFpvfypaTkJiYAULzFs->isSequenceNetwork){ iSoftmaxLayer = 
dwFpvfypaTkJiYAULzFs->network->addSoftMax(*prevLayerTensor); }
#if (NV_TENSORRT_MAJOR >= 5) 
 else{ iSoftmaxLayer = 
dwFpvfypaTkJiYAULzFs->network->addSoftMax(*prevLayerTensor); 
iSoftmaxLayer->setAxes(1<<2); }
#endif
 iSoftmaxLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iSoftmaxLayer->getOutput(0)); } 
MWOutputLayerImpl::MWOutputLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int ) : MWCNNLayerImpl(layer, ntwk_impl) { MWTensor* opTensor = 
getLayer()->getOutputTensor(0); float * QHUGvHzeHXyFElIiOliL; 
CUDA_CALL(hipMalloc((void**)&QHUGvHzeHXyFElIiOliL, sizeof(float) * 
opTensor->getNumElements())); ITensor* prevLayerTensor = getInputITensor(0); 
setOpTensorPtr(prevLayerTensor); opTensor->setData(QHUGvHzeHXyFElIiOliL); } void 
MWOutputLayerImpl::cleanup() { for (int idx = 0; idx < 
aFIIJSbJDJUndqPzwinJ->getNumOutputs(); idx++) { float* data = 
aFIIJSbJDJUndqPzwinJ->getOutputTensor(idx)->getData<float>(); if (data) { 
CUDA_FREE_CALL(data); } } } 
MWAvgPoolingLayerImpl::MWAvgPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int HUlJqbQweWuEmHtzurnN,  int HXjVDkKHrBbbPfJOPKXp,  
int IGBjAMvMJXqrubGDtvyq,  int ILoEtPdZOaUSKUyteZTa,  int GtFSKuDmLreppbjSISoU,  
int GbHRuweETkejIMGyqHDI,  int GmRRxuYauzGdhIlgciAT, int GrowsTaKrpHVUZdgZeJW, 
int ) : MWCNNLayerImpl(layer, ntwk_impl) , iAvgPoolingLayer(0) { ITensor* 
prevLayerTensor = getInputITensor(0); if((GtFSKuDmLreppbjSISoU == 
GbHRuweETkejIMGyqHDI) && (GmRRxuYauzGdhIlgciAT == GrowsTaKrpHVUZdgZeJW)){  
iAvgPoolingLayer = dwFpvfypaTkJiYAULzFs->network->addPooling( *prevLayerTensor, 
PoolingType::kAVERAGE, DimsHW{HUlJqbQweWuEmHtzurnN, HXjVDkKHrBbbPfJOPKXp}); 
iAvgPoolingLayer->setPadding(DimsHW{GtFSKuDmLreppbjSISoU, 
GmRRxuYauzGdhIlgciAT}); } else { IPaddingLayer* iPaddingLayer = 
dwFpvfypaTkJiYAULzFs->network->addPadding( *prevLayerTensor, 
DimsHW{GtFSKuDmLreppbjSISoU,GmRRxuYauzGdhIlgciAT}, 
DimsHW{GbHRuweETkejIMGyqHDI,GrowsTaKrpHVUZdgZeJW}); ITensor* 
FCgChnBQLAUXyoWzTlfg = iPaddingLayer->getOutput(0); iAvgPoolingLayer = 
dwFpvfypaTkJiYAULzFs->network->addPooling( *FCgChnBQLAUXyoWzTlfg, 
PoolingType::kAVERAGE, DimsHW{HUlJqbQweWuEmHtzurnN, HXjVDkKHrBbbPfJOPKXp});  } 
iAvgPoolingLayer->setStride(DimsHW{IGBjAMvMJXqrubGDtvyq, ILoEtPdZOaUSKUyteZTa}); 
iAvgPoolingLayer->setAverageCountExcludesPadding(false); 
iAvgPoolingLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iAvgPoolingLayer->getOutput(0)); }