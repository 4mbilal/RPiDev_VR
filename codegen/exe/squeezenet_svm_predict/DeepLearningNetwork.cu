//
// File: DeepLearningNetwork.cu
//
// GPU Coder version                    : 1.5
// CUDA/C/C++ source code generated on  : 06-Aug-2020 05:06:39
//

// Include Files
#include "DeepLearningNetwork.h"
#include "rt_nonfinite.h"
#include "squeezenet_svm_predict.h"
#include <cstdio>

// Type Definitions
#include "cnn_api.hpp"
#include "MWConvLayer.hpp"
#include "MWDepthConcatenationLayer.hpp"
#include "MWElementwiseAffineLayer.hpp"
#include "MWTargetNetworkImpl.hpp"

// Named Constants
const char * errorString =
  "Abnormal termination due to: %s.\nError in %s (line %d).";

// Function Declarations
static void checkCleanupCudaError(hipError_t errCode, const char * file,
  unsigned int line);

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//

//
// Arguments    : int MaxBufSize
//                int numBufstoAllocate
// Return Type  : void
//

//
// Arguments    : void
// Return Type  : void
//

//
// Arguments    : void
// Return Type  : void
//

//
// Arguments    : MWTargetNetworkImpl *b_targetImpl
//                MWTensor *b
//                int PoolSizeH
//                int PoolSizeW
//                int StrideH
//                int StrideW
//                int PaddingH_T
//                int PaddingH_B
//                int PaddingW_L
//                int PaddingW_R
//                int c
// Return Type  : void
//

//
// Arguments    : MWTargetNetworkImpl *b_targetImpl
//                MWTensor *b
//                int FilterSizeH
//                int FilterSizeW
//                int NumChannels
//                int NumFilters
//                int StrideH
//                int StrideW
//                int PaddingH_Top
//                int PaddingH_Bottom
//                int PaddingW_Left
//                int PaddingW_Right
//                int DilationFactorH
//                int DilationFactorW
//                int NumGroups
//                const char * c_a___codegen_exe_squeezenet_sv
//                const char * d_a___codegen_exe_squeezenet_sv
//                int c
// Return Type  : void
//

//
// Arguments    : MWTargetNetworkImpl *b_targetImpl
//                MWTensor *b
//                int ScaleHeight
//                int ScaleWidth
//                int ScaleChannels
//                int OffsetHeight
//                int OffsetWidth
//                int OffsetChannels
//                bool IsClippedAffine
//                int LowerBound
//                int UpperBound
//                const char * c_a___codegen_exe_squeezenet_sv
//                const char * d_a___codegen_exe_squeezenet_sv
//                int c
// Return Type  : void
//

//
// Arguments    : MWTargetNetworkImpl *b_targetImpl
//                MWTensor *m_in
//                int height
//                int width
//                int channels
//                int withAvg
//                const char * b
//                int c
// Return Type  : void
//

//
// Arguments    : MWTargetNetworkImpl *b_targetImpl
//                MWTensor *b
//                int c
// Return Type  : void
//

//
// Arguments    : MWTargetNetworkImpl *b_targetImpl
//                MWTensor *b
//                int c
// Return Type  : void
//

//
// Arguments    : MWTargetNetworkImpl *b_targetImpl
//                MWTensor *b
//                int inPlaceOp
//                int c
// Return Type  : void
//

//
// Arguments    : MWTargetNetworkImpl *b_targetImpl
//                MWTensor *b
//                int c
// Return Type  : void
//

//
// Arguments    : void
// Return Type  : void
//

//
// Arguments    : void
// Return Type  : void
//

//
// Arguments    : int b
// Return Type  : void
//

//
// Arguments    : void
// Return Type  : float *
//

//
// Arguments    : int handle
// Return Type  : void
//

//
// Arguments    : int b_index
// Return Type  : MWTensor *
//

//
// Arguments    : MWCNNLayer *b_layers[69]
//                int b_numLayers
//                int outputLayerIndices[1]
//                int outputPortIndices[1]
//                int numOutputs
// Return Type  : void
//

//
// Arguments    : void
// Return Type  : void
//

//
// Arguments    : int b_batchSize
// Return Type  : void
//

//
// Arguments    : int b_batchSize
// Return Type  : void
//

//
// Arguments    : int channels
// Return Type  : void
//

//
// Arguments    : float *data
// Return Type  : void
//

//
// Arguments    : int height
// Return Type  : void
//

//
// Arguments    : bool isSequenceNetwork
// Return Type  : void
//

//
// Arguments    : const char * name
// Return Type  : void
//

//
// Arguments    : int sequenceLength
// Return Type  : void
//

//
// Arguments    : int width
// Return Type  : void
//

//
// Arguments    : hipError_t errCode
//                const char * file
//                unsigned int line
// Return Type  : void
//
static void checkCleanupCudaError(hipError_t errCode, const char * file,
  unsigned int line)
{
  if ((errCode != hipSuccess) && (errCode != hipErrorDeinitialized)) {
    printf(errorString, hipGetErrorString(errCode), file, line);
  }
}

//
// Arguments    : int layerIdx
// Return Type  : void
//
void b_squeezenet_0::activations(int)
{
  this->targetImpl->doInference(this->batchSize);
}

//
// Arguments    : void
// Return Type  : void
//
void b_squeezenet_0::allocate()
{
  this->targetImpl->allocate(0, 0);
}

//
// Arguments    : void
// Return Type  : void
//
void b_squeezenet_0::cleanup()
{
  this->deallocate();
  for (int idx = 0; idx < 69; idx++) {
    this->layers[idx]->cleanup();
  }

  if (this->targetImpl) {
    this->targetImpl->cleanup();
  }
}

//
// Arguments    : void
// Return Type  : void
//
void b_squeezenet_0::deallocate()
{
  this->targetImpl->deallocate();
  for (int idx = 0; idx < 69; idx++) {
    this->layers[idx]->deallocate();
  }
}

//
// Arguments    : void
// Return Type  : float *
//
float *b_squeezenet_0::getInputDataPointer()
{
  return this->inputTensor->getFloatData();
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : float *
//
float *b_squeezenet_0::getLayerOutput(int layerIndex, int portIndex)
{
  return this->layers[layerIndex]->getLayerOutput(portIndex);
}

//
// Arguments    : void
// Return Type  : float *
//
float *b_squeezenet_0::getOutputDataPointer()
{
  return this->outputTensor->getFloatData();
}

//
// Arguments    : void
// Return Type  : void
//
void b_squeezenet_0::postsetup()
{
  int activationsIdx[1];
  int activationPortIdx[1];
  activationsIdx[0] = 66;
  activationPortIdx[0] = 0;
  this->targetImpl->postSetup(this->layers, this->numLayers, activationsIdx,
    activationPortIdx, 1);
  for (int idx = 0; idx < 69; idx++) {
    this->layers[idx]->allocate();
  }
}

//
// Arguments    : void
// Return Type  : void
//
void b_squeezenet_0::predict()
{
  this->targetImpl->doInference(this->batchSize);
}

//
// Arguments    : void
// Return Type  : void
//
void b_squeezenet_0::setup()
{
  this->targetImpl->preSetup();
  this->allocate();
  (dynamic_cast<MWInputLayer *>(this->layers[0]))->createInputLayer
    (this->targetImpl, this->inputTensor, 227, 227, 3, 0, "", -1);
  (dynamic_cast<MWElementwiseAffineLayer *>(this->layers[1]))
    ->createElementwiseAffineLayer(this->targetImpl, this->layers[0]
    ->getOutputTensor(0), 1, 1, 3, 1, 1, 3, false, 1, 1,
    "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_data_scale.bin",
    "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_data_offset.bin", -1);
  (dynamic_cast<MWConvLayer *>(this->layers[2]))->createConvLayer
    (this->targetImpl, this->layers[1]->getOutputTensor(0), 3, 3, 3, 64, 2, 2, 0,
     0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_conv1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_conv1_b.bin", -1);
  (dynamic_cast<MWReLULayer *>(this->layers[3]))->createReLULayer
    (this->targetImpl, this->layers[2]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWMaxPoolingLayer *>(this->layers[4]))->createMaxPoolingLayer
    (this->targetImpl, this->layers[3]->getOutputTensor(0), 3, 3, 2, 2, 0, 0, 0,
     0, 0, 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[5]))->createConvLayer
    (this->targetImpl, this->layers[4]->getOutputTensor(0), 1, 1, 64, 16, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire2-squeeze1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire2-squeeze1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[6]))->createReLULayer
    (this->targetImpl, this->layers[5]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[7]))->createConvLayer
    (this->targetImpl, this->layers[6]->getOutputTensor(0), 1, 1, 16, 64, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire2-expand1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire2-expand1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[8]))->createReLULayer
    (this->targetImpl, this->layers[7]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[9]))->createConvLayer
    (this->targetImpl, this->layers[6]->getOutputTensor(0), 3, 3, 16, 64, 1, 1,
     1, 1, 1, 1, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire2-expand3x3_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire2-expand3x3_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[10]))->createReLULayer
    (this->targetImpl, this->layers[9]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWDepthConcatenationLayer *>(this->layers[11]))
    ->createDepthConcatenationLayer(this->targetImpl, 2, this->layers[8]
    ->getOutputTensor(0), this->layers[10]->getOutputTensor(0), -1);
  (dynamic_cast<MWConvLayer *>(this->layers[12]))->createConvLayer
    (this->targetImpl, this->layers[11]->getOutputTensor(0), 1, 1, 128, 16, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire3-squeeze1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire3-squeeze1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[13]))->createReLULayer
    (this->targetImpl, this->layers[12]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[14]))->createConvLayer
    (this->targetImpl, this->layers[13]->getOutputTensor(0), 1, 1, 16, 64, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire3-expand1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire3-expand1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[15]))->createReLULayer
    (this->targetImpl, this->layers[14]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[16]))->createConvLayer
    (this->targetImpl, this->layers[13]->getOutputTensor(0), 3, 3, 16, 64, 1, 1,
     1, 1, 1, 1, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire3-expand3x3_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire3-expand3x3_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[17]))->createReLULayer
    (this->targetImpl, this->layers[16]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWDepthConcatenationLayer *>(this->layers[18]))
    ->createDepthConcatenationLayer(this->targetImpl, 2, this->layers[15]
    ->getOutputTensor(0), this->layers[17]->getOutputTensor(0), -1);
  (dynamic_cast<MWMaxPoolingLayer *>(this->layers[19]))->createMaxPoolingLayer
    (this->targetImpl, this->layers[18]->getOutputTensor(0), 3, 3, 2, 2, 0, 1, 0,
     1, 0, 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[20]))->createConvLayer
    (this->targetImpl, this->layers[19]->getOutputTensor(0), 1, 1, 128, 32, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire4-squeeze1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire4-squeeze1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[21]))->createReLULayer
    (this->targetImpl, this->layers[20]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[22]))->createConvLayer
    (this->targetImpl, this->layers[21]->getOutputTensor(0), 1, 1, 32, 128, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire4-expand1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire4-expand1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[23]))->createReLULayer
    (this->targetImpl, this->layers[22]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[24]))->createConvLayer
    (this->targetImpl, this->layers[21]->getOutputTensor(0), 3, 3, 32, 128, 1, 1,
     1, 1, 1, 1, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire4-expand3x3_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire4-expand3x3_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[25]))->createReLULayer
    (this->targetImpl, this->layers[24]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWDepthConcatenationLayer *>(this->layers[26]))
    ->createDepthConcatenationLayer(this->targetImpl, 2, this->layers[23]
    ->getOutputTensor(0), this->layers[25]->getOutputTensor(0), -1);
  (dynamic_cast<MWConvLayer *>(this->layers[27]))->createConvLayer
    (this->targetImpl, this->layers[26]->getOutputTensor(0), 1, 1, 256, 32, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire5-squeeze1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire5-squeeze1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[28]))->createReLULayer
    (this->targetImpl, this->layers[27]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[29]))->createConvLayer
    (this->targetImpl, this->layers[28]->getOutputTensor(0), 1, 1, 32, 128, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire5-expand1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire5-expand1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[30]))->createReLULayer
    (this->targetImpl, this->layers[29]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[31]))->createConvLayer
    (this->targetImpl, this->layers[28]->getOutputTensor(0), 3, 3, 32, 128, 1, 1,
     1, 1, 1, 1, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire5-expand3x3_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire5-expand3x3_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[32]))->createReLULayer
    (this->targetImpl, this->layers[31]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWDepthConcatenationLayer *>(this->layers[33]))
    ->createDepthConcatenationLayer(this->targetImpl, 2, this->layers[30]
    ->getOutputTensor(0), this->layers[32]->getOutputTensor(0), -1);
  (dynamic_cast<MWMaxPoolingLayer *>(this->layers[34]))->createMaxPoolingLayer
    (this->targetImpl, this->layers[33]->getOutputTensor(0), 3, 3, 2, 2, 0, 1, 0,
     1, 0, 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[35]))->createConvLayer
    (this->targetImpl, this->layers[34]->getOutputTensor(0), 1, 1, 256, 48, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire6-squeeze1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire6-squeeze1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[36]))->createReLULayer
    (this->targetImpl, this->layers[35]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[37]))->createConvLayer
    (this->targetImpl, this->layers[36]->getOutputTensor(0), 1, 1, 48, 192, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire6-expand1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire6-expand1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[38]))->createReLULayer
    (this->targetImpl, this->layers[37]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[39]))->createConvLayer
    (this->targetImpl, this->layers[36]->getOutputTensor(0), 3, 3, 48, 192, 1, 1,
     1, 1, 1, 1, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire6-expand3x3_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire6-expand3x3_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[40]))->createReLULayer
    (this->targetImpl, this->layers[39]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWDepthConcatenationLayer *>(this->layers[41]))
    ->createDepthConcatenationLayer(this->targetImpl, 2, this->layers[38]
    ->getOutputTensor(0), this->layers[40]->getOutputTensor(0), -1);
  (dynamic_cast<MWConvLayer *>(this->layers[42]))->createConvLayer
    (this->targetImpl, this->layers[41]->getOutputTensor(0), 1, 1, 384, 48, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire7-squeeze1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire7-squeeze1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[43]))->createReLULayer
    (this->targetImpl, this->layers[42]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[44]))->createConvLayer
    (this->targetImpl, this->layers[43]->getOutputTensor(0), 1, 1, 48, 192, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire7-expand1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire7-expand1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[45]))->createReLULayer
    (this->targetImpl, this->layers[44]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[46]))->createConvLayer
    (this->targetImpl, this->layers[43]->getOutputTensor(0), 3, 3, 48, 192, 1, 1,
     1, 1, 1, 1, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire7-expand3x3_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire7-expand3x3_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[47]))->createReLULayer
    (this->targetImpl, this->layers[46]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWDepthConcatenationLayer *>(this->layers[48]))
    ->createDepthConcatenationLayer(this->targetImpl, 2, this->layers[45]
    ->getOutputTensor(0), this->layers[47]->getOutputTensor(0), -1);
  (dynamic_cast<MWConvLayer *>(this->layers[49]))->createConvLayer
    (this->targetImpl, this->layers[48]->getOutputTensor(0), 1, 1, 384, 64, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire8-squeeze1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire8-squeeze1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[50]))->createReLULayer
    (this->targetImpl, this->layers[49]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[51]))->createConvLayer
    (this->targetImpl, this->layers[50]->getOutputTensor(0), 1, 1, 64, 256, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire8-expand1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire8-expand1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[52]))->createReLULayer
    (this->targetImpl, this->layers[51]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[53]))->createConvLayer
    (this->targetImpl, this->layers[50]->getOutputTensor(0), 3, 3, 64, 256, 1, 1,
     1, 1, 1, 1, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire8-expand3x3_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire8-expand3x3_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[54]))->createReLULayer
    (this->targetImpl, this->layers[53]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWDepthConcatenationLayer *>(this->layers[55]))
    ->createDepthConcatenationLayer(this->targetImpl, 2, this->layers[52]
    ->getOutputTensor(0), this->layers[54]->getOutputTensor(0), -1);
  (dynamic_cast<MWConvLayer *>(this->layers[56]))->createConvLayer
    (this->targetImpl, this->layers[55]->getOutputTensor(0), 1, 1, 512, 64, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire9-squeeze1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire9-squeeze1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[57]))->createReLULayer
    (this->targetImpl, this->layers[56]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[58]))->createConvLayer
    (this->targetImpl, this->layers[57]->getOutputTensor(0), 1, 1, 64, 256, 1, 1,
     0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire9-expand1x1_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire9-expand1x1_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[59]))->createReLULayer
    (this->targetImpl, this->layers[58]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWConvLayer *>(this->layers[60]))->createConvLayer
    (this->targetImpl, this->layers[57]->getOutputTensor(0), 3, 3, 64, 256, 1, 1,
     1, 1, 1, 1, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire9-expand3x3_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_fire9-expand3x3_b.bin",
     -1);
  (dynamic_cast<MWReLULayer *>(this->layers[61]))->createReLULayer
    (this->targetImpl, this->layers[60]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWDepthConcatenationLayer *>(this->layers[62]))
    ->createDepthConcatenationLayer(this->targetImpl, 2, this->layers[59]
    ->getOutputTensor(0), this->layers[61]->getOutputTensor(0), -1);
  (dynamic_cast<MWPassthroughLayer *>(this->layers[63]))->createPassthroughLayer
    (this->targetImpl, this->layers[62]->getOutputTensor(0), -1);
  (dynamic_cast<MWConvLayer *>(this->layers[64]))->createConvLayer
    (this->targetImpl, this->layers[63]->getOutputTensor(0), 1, 1, 512, 1000, 1,
     1, 0, 0, 0, 0, 1, 1, 1,
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_conv10_w.bin",
     "./codegen/exe/squeezenet_svm_predict/cnn_squeezenet_conv10_b.bin", -1);
  (dynamic_cast<MWReLULayer *>(this->layers[65]))->createReLULayer
    (this->targetImpl, this->layers[64]->getOutputTensor(0), 1, -1);
  (dynamic_cast<MWAvgPoolingLayer *>(this->layers[66]))->createAvgPoolingLayer
    (this->targetImpl, this->layers[65]->getOutputTensor(0), -1, -1, 1, 1, 0, 0,
     0, 0, -1);
  (dynamic_cast<MWSoftmaxLayer *>(this->layers[67]))->createSoftmaxLayer
    (this->targetImpl, this->layers[66]->getOutputTensor(0), -1);
  (dynamic_cast<MWOutputLayer *>(this->layers[68]))->createOutputLayer
    (this->targetImpl, this->layers[67]->getOutputTensor(0), -1);
  this->postsetup();
  this->inputTensor->setData(this->layers[0]->getLayerOutput(0));
  this->outputTensor->setData(this->layers[68]->getLayerOutput(0));
  this->inputData = this->inputTensor->getFloatData();
  this->outputData = this->outputTensor->getFloatData();
}

//
// Arguments    : void
// Return Type  : void
//
b_squeezenet_0::~b_squeezenet_0()
{
  this->cleanup();
  checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
  for (int idx = 0; idx < 69; idx++) {
    delete this->layers[idx];
  }

  if (this->targetImpl) {
    delete this->targetImpl;
  }

  delete this->inputTensor;
  delete this->outputTensor;
}

//
// Arguments    : void
// Return Type  : void
//
b_squeezenet_0::b_squeezenet_0()
{
  this->numLayers = 69;
  this->targetImpl = 0;
  this->layers[0] = new MWInputLayer;
  this->layers[0]->setName("data");
  this->layers[1] = new MWElementwiseAffineLayer;
  this->layers[1]->setName("data_normalization");
  this->layers[2] = new MWConvLayer;
  this->layers[2]->setName("conv1");
  this->layers[3] = new MWReLULayer;
  this->layers[3]->setName("relu_conv1");
  this->layers[4] = new MWMaxPoolingLayer;
  this->layers[4]->setName("pool1");
  this->layers[5] = new MWConvLayer;
  this->layers[5]->setName("fire2-squeeze1x1");
  this->layers[6] = new MWReLULayer;
  this->layers[6]->setName("fire2-relu_squeeze1x1");
  this->layers[7] = new MWConvLayer;
  this->layers[7]->setName("fire2-expand1x1");
  this->layers[8] = new MWReLULayer;
  this->layers[8]->setName("fire2-relu_expand1x1");
  this->layers[9] = new MWConvLayer;
  this->layers[9]->setName("fire2-expand3x3");
  this->layers[10] = new MWReLULayer;
  this->layers[10]->setName("fire2-relu_expand3x3");
  this->layers[11] = new MWDepthConcatenationLayer;
  this->layers[11]->setName("fire2-concat");
  this->layers[12] = new MWConvLayer;
  this->layers[12]->setName("fire3-squeeze1x1");
  this->layers[13] = new MWReLULayer;
  this->layers[13]->setName("fire3-relu_squeeze1x1");
  this->layers[14] = new MWConvLayer;
  this->layers[14]->setName("fire3-expand1x1");
  this->layers[15] = new MWReLULayer;
  this->layers[15]->setName("fire3-relu_expand1x1");
  this->layers[16] = new MWConvLayer;
  this->layers[16]->setName("fire3-expand3x3");
  this->layers[17] = new MWReLULayer;
  this->layers[17]->setName("fire3-relu_expand3x3");
  this->layers[18] = new MWDepthConcatenationLayer;
  this->layers[18]->setName("fire3-concat");
  this->layers[19] = new MWMaxPoolingLayer;
  this->layers[19]->setName("pool3");
  this->layers[20] = new MWConvLayer;
  this->layers[20]->setName("fire4-squeeze1x1");
  this->layers[21] = new MWReLULayer;
  this->layers[21]->setName("fire4-relu_squeeze1x1");
  this->layers[22] = new MWConvLayer;
  this->layers[22]->setName("fire4-expand1x1");
  this->layers[23] = new MWReLULayer;
  this->layers[23]->setName("fire4-relu_expand1x1");
  this->layers[24] = new MWConvLayer;
  this->layers[24]->setName("fire4-expand3x3");
  this->layers[25] = new MWReLULayer;
  this->layers[25]->setName("fire4-relu_expand3x3");
  this->layers[26] = new MWDepthConcatenationLayer;
  this->layers[26]->setName("fire4-concat");
  this->layers[27] = new MWConvLayer;
  this->layers[27]->setName("fire5-squeeze1x1");
  this->layers[28] = new MWReLULayer;
  this->layers[28]->setName("fire5-relu_squeeze1x1");
  this->layers[29] = new MWConvLayer;
  this->layers[29]->setName("fire5-expand1x1");
  this->layers[30] = new MWReLULayer;
  this->layers[30]->setName("fire5-relu_expand1x1");
  this->layers[31] = new MWConvLayer;
  this->layers[31]->setName("fire5-expand3x3");
  this->layers[32] = new MWReLULayer;
  this->layers[32]->setName("fire5-relu_expand3x3");
  this->layers[33] = new MWDepthConcatenationLayer;
  this->layers[33]->setName("fire5-concat");
  this->layers[34] = new MWMaxPoolingLayer;
  this->layers[34]->setName("pool5");
  this->layers[35] = new MWConvLayer;
  this->layers[35]->setName("fire6-squeeze1x1");
  this->layers[36] = new MWReLULayer;
  this->layers[36]->setName("fire6-relu_squeeze1x1");
  this->layers[37] = new MWConvLayer;
  this->layers[37]->setName("fire6-expand1x1");
  this->layers[38] = new MWReLULayer;
  this->layers[38]->setName("fire6-relu_expand1x1");
  this->layers[39] = new MWConvLayer;
  this->layers[39]->setName("fire6-expand3x3");
  this->layers[40] = new MWReLULayer;
  this->layers[40]->setName("fire6-relu_expand3x3");
  this->layers[41] = new MWDepthConcatenationLayer;
  this->layers[41]->setName("fire6-concat");
  this->layers[42] = new MWConvLayer;
  this->layers[42]->setName("fire7-squeeze1x1");
  this->layers[43] = new MWReLULayer;
  this->layers[43]->setName("fire7-relu_squeeze1x1");
  this->layers[44] = new MWConvLayer;
  this->layers[44]->setName("fire7-expand1x1");
  this->layers[45] = new MWReLULayer;
  this->layers[45]->setName("fire7-relu_expand1x1");
  this->layers[46] = new MWConvLayer;
  this->layers[46]->setName("fire7-expand3x3");
  this->layers[47] = new MWReLULayer;
  this->layers[47]->setName("fire7-relu_expand3x3");
  this->layers[48] = new MWDepthConcatenationLayer;
  this->layers[48]->setName("fire7-concat");
  this->layers[49] = new MWConvLayer;
  this->layers[49]->setName("fire8-squeeze1x1");
  this->layers[50] = new MWReLULayer;
  this->layers[50]->setName("fire8-relu_squeeze1x1");
  this->layers[51] = new MWConvLayer;
  this->layers[51]->setName("fire8-expand1x1");
  this->layers[52] = new MWReLULayer;
  this->layers[52]->setName("fire8-relu_expand1x1");
  this->layers[53] = new MWConvLayer;
  this->layers[53]->setName("fire8-expand3x3");
  this->layers[54] = new MWReLULayer;
  this->layers[54]->setName("fire8-relu_expand3x3");
  this->layers[55] = new MWDepthConcatenationLayer;
  this->layers[55]->setName("fire8-concat");
  this->layers[56] = new MWConvLayer;
  this->layers[56]->setName("fire9-squeeze1x1");
  this->layers[57] = new MWReLULayer;
  this->layers[57]->setName("fire9-relu_squeeze1x1");
  this->layers[58] = new MWConvLayer;
  this->layers[58]->setName("fire9-expand1x1");
  this->layers[59] = new MWReLULayer;
  this->layers[59]->setName("fire9-relu_expand1x1");
  this->layers[60] = new MWConvLayer;
  this->layers[60]->setName("fire9-expand3x3");
  this->layers[61] = new MWReLULayer;
  this->layers[61]->setName("fire9-relu_expand3x3");
  this->layers[62] = new MWDepthConcatenationLayer;
  this->layers[62]->setName("fire9-concat");
  this->layers[63] = new MWPassthroughLayer;
  this->layers[63]->setName("drop9");
  this->layers[64] = new MWConvLayer;
  this->layers[64]->setName("conv10");
  this->layers[65] = new MWReLULayer;
  this->layers[65]->setName("relu_conv10");
  this->layers[66] = new MWAvgPoolingLayer;
  this->layers[66]->setName("pool10");
  this->layers[67] = new MWSoftmaxLayer;
  this->layers[67]->setName("prob");
  this->layers[68] = new MWOutputLayer;
  this->layers[68]->setName("ClassificationLayer_predictions");
  this->targetImpl = new MWTargetNetworkImpl;
  this->targetImpl->setBatchSize(1);
  this->targetImpl->setIsSequenceNetwork(false);
  this->inputTensor = new MWTensor;
  this->inputTensor->setHeight(227);
  this->inputTensor->setWidth(227);
  this->inputTensor->setChannels(3);
  this->inputTensor->setBatchSize(1);
  this->inputTensor->setSequenceLength(1);
  this->outputTensor = new MWTensor;
}

//
// Arguments    : b_squeezenet_0 *obj
// Return Type  : void
//
void DeepLearningNetwork_setup(b_squeezenet_0 *obj)
{
  obj->setup();
  obj->batchSize = 1;
}

//
// File trailer for DeepLearningNetwork.cu
//
// [EOF]
//
