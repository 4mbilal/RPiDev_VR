//
// File: squeezenet_svm_predict_terminate.cu
//
// GPU Coder version                    : 1.5
// CUDA/C/C++ source code generated on  : 06-Aug-2020 05:06:39
//

// Include Files
#include "squeezenet_svm_predict_terminate.h"
#include "DeepLearningNetwork.h"
#include "rt_nonfinite.h"
#include "squeezenet_svm_predict.h"
#include "squeezenet_svm_predict_data.h"

// Function Declarations
static void cublasEnsureDestruction();

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//
static void cublasEnsureDestruction()
{
  if (cublasGlobalHandle != NULL) {
    hipblasDestroy(cublasGlobalHandle);
    cublasGlobalHandle = NULL;
  }
}

//
// Arguments    : void
// Return Type  : void
//
void squeezenet_svm_predict_terminate()
{
  cublasEnsureDestruction();
  isInitialized_squeezenet_svm_predict = false;
}

//
// File trailer for squeezenet_svm_predict_terminate.cu
//
// [EOF]
//
