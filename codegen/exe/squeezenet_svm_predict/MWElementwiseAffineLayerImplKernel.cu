#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <stdio.h>
 void __global__ __launch_bounds__(1024) scale_scalar_kernel(float* 
inputBuffer, float* outputBuffer, float* omxlPZbBePZdWaJOBUUG, long int 
YGiQICncmsGZkNUyiQyg) {  for (int idx = blockDim.x * blockIdx.x + threadIdx.x; 
idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) {  outputBuffer[idx] 
= omxlPZbBePZdWaJOBUUG[0]*inputBuffer[idx]; } } void __global__ 
__launch_bounds__(1024) scale_vector_kernel(float* inputBuffer, float* 
outputBuffer, float* omxlPZbBePZdWaJOBUUG, long int YNmJhGSUszJKxsodxiuV, 
long int YNDVziqpDddiXQKYZZhX, long int YGiQICncmsGZkNUyiQyg) {  for 
(int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; 
idx += blockDim.x * gridDim.x) { int dAGMlbhOYuZqhuDGCqih = 
idx/YNDVziqpDddiXQKYZZhX; long int EfvWctmlsWAPsxXgdKWf = 
idx-(YNDVziqpDddiXQKYZZhX*dAGMlbhOYuZqhuDGCqih); int KHjdvykTFbUxdfZTFbqy = 
static_cast<int>(EfvWctmlsWAPsxXgdKWf / YNmJhGSUszJKxsodxiuV); 
outputBuffer[idx] = omxlPZbBePZdWaJOBUUG[KHjdvykTFbUxdfZTFbqy]*inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) scale_matrix2d_kernel(float* inputBuffer, 
float* outputBuffer, float* omxlPZbBePZdWaJOBUUG, long int YOWMnLKOMqAODXiVNoGy, long 
int YNmJhGSUszJKxsodxiuV, long int YNDVziqpDddiXQKYZZhX, long 
int YGiQICncmsGZkNUyiQyg) {  for (int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { int 
dAGMlbhOYuZqhuDGCqih = idx/YNDVziqpDddiXQKYZZhX; long int EfvWctmlsWAPsxXgdKWf 
= idx-(YNDVziqpDddiXQKYZZhX*dAGMlbhOYuZqhuDGCqih); int KHjdvykTFbUxdfZTFbqy = 
static_cast<int>(EfvWctmlsWAPsxXgdKWf / YNmJhGSUszJKxsodxiuV); long 
int EvebzoroiuKkIxwjkGnD = EfvWctmlsWAPsxXgdKWf - 
(YNmJhGSUszJKxsodxiuV*KHjdvykTFbUxdfZTFbqy); int vIWQzNvYZSuxmOTVDFhU = 
static_cast<int>(EvebzoroiuKkIxwjkGnD % YOWMnLKOMqAODXiVNoGy); int 
RAtlBpdedvgxUsgDTsch = static_cast<int>(EvebzoroiuKkIxwjkGnD / YOWMnLKOMqAODXiVNoGy); 
outputBuffer[idx] = 
omxlPZbBePZdWaJOBUUG[vIWQzNvYZSuxmOTVDFhU+YOWMnLKOMqAODXiVNoGy*RAtlBpdedvgxUsgDTsch]*inputBuffer[idx]; 
} } void __global__ __launch_bounds__(1024) scale_tensor3d_kernel(float* 
inputBuffer, float* outputBuffer, float* omxlPZbBePZdWaJOBUUG, long int 
YOWMnLKOMqAODXiVNoGy, long int YCSvyQZBWMDYQXHtyVai, long int 
YNmJhGSUszJKxsodxiuV, long int YNDVziqpDddiXQKYZZhX, long int 
YGiQICncmsGZkNUyiQyg) {  for (int idx = blockDim.x * blockIdx.x + threadIdx.x; 
idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { int dAGMlbhOYuZqhuDGCqih = 
idx/YNDVziqpDddiXQKYZZhX; long int EfvWctmlsWAPsxXgdKWf = 
idx-(YNDVziqpDddiXQKYZZhX*dAGMlbhOYuZqhuDGCqih); int KHjdvykTFbUxdfZTFbqy = 
static_cast<int>(EfvWctmlsWAPsxXgdKWf / YNmJhGSUszJKxsodxiuV); long 
int EvebzoroiuKkIxwjkGnD = EfvWctmlsWAPsxXgdKWf - 
(YNmJhGSUszJKxsodxiuV*KHjdvykTFbUxdfZTFbqy); int vIWQzNvYZSuxmOTVDFhU = 
static_cast<int>(EvebzoroiuKkIxwjkGnD % YOWMnLKOMqAODXiVNoGy); int 
RAtlBpdedvgxUsgDTsch = static_cast<int>(EvebzoroiuKkIxwjkGnD / YOWMnLKOMqAODXiVNoGy); 
outputBuffer[idx] = 
omxlPZbBePZdWaJOBUUG[vIWQzNvYZSuxmOTVDFhU+YOWMnLKOMqAODXiVNoGy*(RAtlBpdedvgxUsgDTsch+YCSvyQZBWMDYQXHtyVai*KHjdvykTFbUxdfZTFbqy)]*inputBuffer[idx]; 
} }  void __global__ __launch_bounds__(1024) offset_scalar_kernel(float* 
inputBuffer, float* outputBuffer, float* gCYwEfkibolsgZAumsuW, long int 
YGiQICncmsGZkNUyiQyg, bool ZKjSVYDDjACizBkGbqBq, int bOrQjJTNlssnrexxbHdi, int 
unSXtdjDjpysqxmbIiPv) {  for (int idx = blockDim.x * blockIdx.x + threadIdx.x; 
idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { outputBuffer[idx] 
= inputBuffer[idx] + gCYwEfkibolsgZAumsuW[0]; if (ZKjSVYDDjACizBkGbqBq){ 
outputBuffer[idx] = outputBuffer[idx] > unSXtdjDjpysqxmbIiPv ? 
unSXtdjDjpysqxmbIiPv : outputBuffer[idx]; outputBuffer[idx] = 
outputBuffer[idx] < bOrQjJTNlssnrexxbHdi ? bOrQjJTNlssnrexxbHdi : 
outputBuffer[idx]; } } } void __global__ __launch_bounds__(1024) 
offset_vector_kernel(float* inputBuffer, float* outputBuffer, float* 
gCYwEfkibolsgZAumsuW,  long int YNmJhGSUszJKxsodxiuV, long int 
YNDVziqpDddiXQKYZZhX, long int YGiQICncmsGZkNUyiQyg, bool 
ZKjSVYDDjACizBkGbqBq, int bOrQjJTNlssnrexxbHdi, int unSXtdjDjpysqxmbIiPv) {  
for (int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { int dAGMlbhOYuZqhuDGCqih = 
idx/YNDVziqpDddiXQKYZZhX; long int EfvWctmlsWAPsxXgdKWf = 
idx-(YNDVziqpDddiXQKYZZhX*dAGMlbhOYuZqhuDGCqih); int KHjdvykTFbUxdfZTFbqy = 
static_cast<int>(EfvWctmlsWAPsxXgdKWf / YNmJhGSUszJKxsodxiuV); 
outputBuffer[idx] = inputBuffer[idx] + gCYwEfkibolsgZAumsuW[KHjdvykTFbUxdfZTFbqy]; if 
(ZKjSVYDDjACizBkGbqBq){ outputBuffer[idx] = outputBuffer[idx] > 
unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : outputBuffer[idx]; 
outputBuffer[idx] = outputBuffer[idx] < bOrQjJTNlssnrexxbHdi ? 
bOrQjJTNlssnrexxbHdi : outputBuffer[idx]; } } } void __global__ 
__launch_bounds__(1024) offset_matrix2d_kernel(float* inputBuffer, float* 
outputBuffer, float* gCYwEfkibolsgZAumsuW, long int YOWMnLKOMqAODXiVNoGy, long int 
YNmJhGSUszJKxsodxiuV, long int YNDVziqpDddiXQKYZZhX, long int 
YGiQICncmsGZkNUyiQyg, bool ZKjSVYDDjACizBkGbqBq, int bOrQjJTNlssnrexxbHdi, int 
unSXtdjDjpysqxmbIiPv) {  for (int idx = blockDim.x * blockIdx.x + threadIdx.x; 
idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { int dAGMlbhOYuZqhuDGCqih = 
idx/YNDVziqpDddiXQKYZZhX; long int EfvWctmlsWAPsxXgdKWf = 
idx-(YNDVziqpDddiXQKYZZhX*dAGMlbhOYuZqhuDGCqih); int KHjdvykTFbUxdfZTFbqy = 
static_cast<int>(EfvWctmlsWAPsxXgdKWf / YNmJhGSUszJKxsodxiuV); long 
int EvebzoroiuKkIxwjkGnD = EfvWctmlsWAPsxXgdKWf - 
(YNmJhGSUszJKxsodxiuV*KHjdvykTFbUxdfZTFbqy); int vIWQzNvYZSuxmOTVDFhU = 
static_cast<int>(EvebzoroiuKkIxwjkGnD % YOWMnLKOMqAODXiVNoGy); int 
RAtlBpdedvgxUsgDTsch = static_cast<int>(EvebzoroiuKkIxwjkGnD / YOWMnLKOMqAODXiVNoGy); 
outputBuffer[idx] = inputBuffer[idx] + 
gCYwEfkibolsgZAumsuW[vIWQzNvYZSuxmOTVDFhU+YOWMnLKOMqAODXiVNoGy*RAtlBpdedvgxUsgDTsch]; if 
(ZKjSVYDDjACizBkGbqBq){ outputBuffer[idx] = outputBuffer[idx] > 
unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : outputBuffer[idx]; 
outputBuffer[idx] = outputBuffer[idx] < bOrQjJTNlssnrexxbHdi ? 
bOrQjJTNlssnrexxbHdi : outputBuffer[idx]; } } } void __global__ 
__launch_bounds__(1024) offset_tensor3d_kernel(float* inputBuffer, float* 
outputBuffer, float* gCYwEfkibolsgZAumsuW,  long int YOWMnLKOMqAODXiVNoGy, long int 
YCSvyQZBWMDYQXHtyVai, long int YNmJhGSUszJKxsodxiuV, long int 
YNDVziqpDddiXQKYZZhX, long int YGiQICncmsGZkNUyiQyg, bool 
ZKjSVYDDjACizBkGbqBq, int bOrQjJTNlssnrexxbHdi, int unSXtdjDjpysqxmbIiPv) {  
for (int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { int dAGMlbhOYuZqhuDGCqih = 
idx/YNDVziqpDddiXQKYZZhX; long int EfvWctmlsWAPsxXgdKWf = 
idx-(YNDVziqpDddiXQKYZZhX*dAGMlbhOYuZqhuDGCqih); int KHjdvykTFbUxdfZTFbqy = 
static_cast<int>(EfvWctmlsWAPsxXgdKWf / YNmJhGSUszJKxsodxiuV); long 
int EvebzoroiuKkIxwjkGnD = EfvWctmlsWAPsxXgdKWf - 
(YNmJhGSUszJKxsodxiuV*KHjdvykTFbUxdfZTFbqy); int vIWQzNvYZSuxmOTVDFhU = 
static_cast<int>(EvebzoroiuKkIxwjkGnD % YOWMnLKOMqAODXiVNoGy); int 
RAtlBpdedvgxUsgDTsch = static_cast<int>(EvebzoroiuKkIxwjkGnD / YOWMnLKOMqAODXiVNoGy); 
outputBuffer[idx] = inputBuffer[idx] + 
gCYwEfkibolsgZAumsuW[vIWQzNvYZSuxmOTVDFhU+YOWMnLKOMqAODXiVNoGy*(RAtlBpdedvgxUsgDTsch+YCSvyQZBWMDYQXHtyVai*KHjdvykTFbUxdfZTFbqy)]; 
if (ZKjSVYDDjACizBkGbqBq){ outputBuffer[idx] = outputBuffer[idx] > 
unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : outputBuffer[idx]; 
outputBuffer[idx] = outputBuffer[idx] < bOrQjJTNlssnrexxbHdi ? 
bOrQjJTNlssnrexxbHdi : outputBuffer[idx]; } } } 