#include "MWElementwiseAffineLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWKernelHeaders.hpp"
 MWElementwiseAffineLayerImpl::MWElementwiseAffineLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int scale_H,  int scale_W,  int scale_C, int 
offset_H,  int offset_W,  int offset_C, bool isClipped,  int lowerbound,  int 
upperbound, const char* scale_file,  const char* offset_file, int ) : 
MWCNNLayerImpl(layer, ntwk_impl), pxmnUEWGnfCxJNuDkXAo(NULL), 
hTbrmZSPUORqMIprVbKk(NULL), rZyMIPooLjRiXLgSWDuw(scale_H), 
rfIckzovFcUSejaEKNlY(scale_W), qquNiJHQtfSLDMNCPIBJ(scale_C), 
hpOzCTZasBMYKoXVxMDZ(offset_H), hqbKXLMjsDxRQqyJEgbg(offset_W), 
hjXpmIaeqlKKSDDBSxtE(offset_C), ZqQxEyCjEixByRZYMkbj(isClipped), 
boZxhUbgnNUlJQXTuWwi(lowerbound), voqEJSkAwmNPuqzoiuom(upperbound), 
rwPhFWHcKnJsClVtebGW(nullptr), ikTyjLTPRBkBRlLSyxXG(nullptr), 
qJWXFXvcpbSwehmlTNru(0), HKStLBswJlYYprZPPGQx(0), mJHKRLbEVeCWkBgMqjYL(0) { 
loadScaleAndOffset(scale_file, offset_file); setLayerProperties(); bool 
isMatrix2d = (rZyMIPooLjRiXLgSWDuw > 1) && (rfIckzovFcUSejaEKNlY > 1) && 
(qquNiJHQtfSLDMNCPIBJ != WShIchjHSSzOoYZernlN); if ((!ZqQxEyCjEixByRZYMkbj) && 
(reGtUwUlPSwEenEBVIzH == hqVFaqkobRNLQNgtbaai ) && !isMatrix2d && 
(!dwFpvfypaTkJiYAULzFs->isSequenceNetwork)) { qeQuIDaHqnxGPDbPoQJF.values 
= rwPhFWHcKnJsClVtebGW; qeQuIDaHqnxGPDbPoQJF.count = reGtUwUlPSwEenEBVIzH; 
qeQuIDaHqnxGPDbPoQJF.type = DataType::kFLOAT; 
niGnnRufksTFnsUUxnCj.values = nullptr; niGnnRufksTFnsUUxnCj.count = 
0; niGnnRufksTFnsUUxnCj.type = DataType::kFLOAT; 
suFVgcuEVpCOrewbJfkB.values = ikTyjLTPRBkBRlLSyxXG; 
suFVgcuEVpCOrewbJfkB.count = hqVFaqkobRNLQNgtbaai; 
suFVgcuEVpCOrewbJfkB.type = DataType::kFLOAT; ITensor* prevLayerTensor = 
getInputITensor(0); ScaleMode mode; if (reGtUwUlPSwEenEBVIzH == 1) mode = 
ScaleMode::kUNIFORM; else if (YMNbgnUYZspjMLjwcIOS == 
reGtUwUlPSwEenEBVIzH) mode = ScaleMode::kELEMENTWISE; else if (rZyMIPooLjRiXLgSWDuw 
== 1 && rfIckzovFcUSejaEKNlY == 1 && reGtUwUlPSwEenEBVIzH == qquNiJHQtfSLDMNCPIBJ) 
mode = ScaleMode::kCHANNEL; qJWXFXvcpbSwehmlTNru = 
dwFpvfypaTkJiYAULzFs->network->addScale(*prevLayerTensor,  mode,  
suFVgcuEVpCOrewbJfkB, qeQuIDaHqnxGPDbPoQJF,  
niGnnRufksTFnsUUxnCj); assert(qJWXFXvcpbSwehmlTNru); 
qJWXFXvcpbSwehmlTNru->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(qJWXFXvcpbSwehmlTNru->getOutput(0)); } else { ITensor* 
prevLayerTensor = getInputITensor(0); mJHKRLbEVeCWkBgMqjYL = new 
MWPluginInterfaceImpl(this); HKStLBswJlYYprZPPGQx = 
dwFpvfypaTkJiYAULzFs->network->addPlugin(&prevLayerTensor, 1, 
*mJHKRLbEVeCWkBgMqjYL); 
HKStLBswJlYYprZPPGQx->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(HKStLBswJlYYprZPPGQx->getOutput(0)); } } void 
MWElementwiseAffineLayerImpl::loadScaleAndOffset(const char* 
sCDdEyIOjXBVHhcakBhd, const char* jLmklYtHcmTxayQTpmRw){ 
CUDA_CALL(hipMalloc((void**)&pxmnUEWGnfCxJNuDkXAo, 
sizeof(float)*rZyMIPooLjRiXLgSWDuw*rfIckzovFcUSejaEKNlY*qquNiJHQtfSLDMNCPIBJ)); 
CUDA_CALL(hipMalloc((void**)&hTbrmZSPUORqMIprVbKk, 
sizeof(float)*hpOzCTZasBMYKoXVxMDZ*hqbKXLMjsDxRQqyJEgbg*hjXpmIaeqlKKSDDBSxtE));  
loadScale(sCDdEyIOjXBVHhcakBhd); loadOffset(jLmklYtHcmTxayQTpmRw); } void 
MWElementwiseAffineLayerImpl::setLayerProperties(){ WawamKKnqecNqBXIyHIl = 
getLayer()->getInputTensor(0)->getHeight(); WbTBQxsNsCURmwRhNTAD = 
getLayer()->getInputTensor(0)->getWidth(); WShIchjHSSzOoYZernlN = 
getLayer()->getInputTensor(0)->getChannels(); YmfPcXPXNFZDznkzKZrl = 
WawamKKnqecNqBXIyHIl*WbTBQxsNsCURmwRhNTAD; YMNbgnUYZspjMLjwcIOS = 
YmfPcXPXNFZDznkzKZrl*WShIchjHSSzOoYZernlN; YDoginwuwFxabuYCVqpT = 
getLayer()->getInputTensor(0)->getNumElements(); reGtUwUlPSwEenEBVIzH = 
rZyMIPooLjRiXLgSWDuw * rfIckzovFcUSejaEKNlY * qquNiJHQtfSLDMNCPIBJ; 
hqVFaqkobRNLQNgtbaai = hpOzCTZasBMYKoXVxMDZ * hqbKXLMjsDxRQqyJEgbg * 
hjXpmIaeqlKKSDDBSxtE; assert(reGtUwUlPSwEenEBVIzH <= YDoginwuwFxabuYCVqpT); 
assert(hqVFaqkobRNLQNgtbaai <= YDoginwuwFxabuYCVqpT); } int 
MWElementwiseAffineLayerImpl::pluginEnqueueImpl(const void* const* inputs, 
void** outputs) { long int uTUuLVVebDakbPjXOQwp = ((YDoginwuwFxabuYCVqpT + 31) / 32) 
* 32; long int vVyVzWKKaCvGClCSagOb = (uTUuLVVebDakbPjXOQwp < 1024) ? 
uTUuLVVebDakbPjXOQwp : 1024; long int OuTwywxKeMgznElXdjGp = 
(YDoginwuwFxabuYCVqpT + vVyVzWKKaCvGClCSagOb - 1) / 
vVyVzWKKaCvGClCSagOb; if (reGtUwUlPSwEenEBVIzH == 1) { 
scale_scalar_kernel<<<OuTwywxKeMgznElXdjGp, vVyVzWKKaCvGClCSagOb>>>( 
(float*)inputs[0],  (float*)outputs[0], pxmnUEWGnfCxJNuDkXAo, 
YDoginwuwFxabuYCVqpT); } else if (rZyMIPooLjRiXLgSWDuw == 1 && rfIckzovFcUSejaEKNlY 
== 1 && reGtUwUlPSwEenEBVIzH > 1) { 
scale_vector_kernel<<<OuTwywxKeMgznElXdjGp, vVyVzWKKaCvGClCSagOb>>>( 
(float*)inputs[0],  (float*)outputs[0], pxmnUEWGnfCxJNuDkXAo, 
YmfPcXPXNFZDznkzKZrl, YMNbgnUYZspjMLjwcIOS, 
YDoginwuwFxabuYCVqpT); } else if (YMNbgnUYZspjMLjwcIOS == 
reGtUwUlPSwEenEBVIzH) {  scale_tensor3d_kernel<<<OuTwywxKeMgznElXdjGp, 
vVyVzWKKaCvGClCSagOb>>>( (float*)inputs[0],  (float*)outputs[0], 
pxmnUEWGnfCxJNuDkXAo, WbTBQxsNsCURmwRhNTAD, WawamKKnqecNqBXIyHIl,  
YmfPcXPXNFZDznkzKZrl,  YMNbgnUYZspjMLjwcIOS, 
YDoginwuwFxabuYCVqpT); } else { 
scale_matrix2d_kernel<<<OuTwywxKeMgznElXdjGp, 
vVyVzWKKaCvGClCSagOb>>>( (float*)inputs[0],  (float*)outputs[0], 
pxmnUEWGnfCxJNuDkXAo, WbTBQxsNsCURmwRhNTAD,  YmfPcXPXNFZDznkzKZrl,  
YMNbgnUYZspjMLjwcIOS, YDoginwuwFxabuYCVqpT); } if (hqVFaqkobRNLQNgtbaai 
== 1) { offset_scalar_kernel<<<OuTwywxKeMgznElXdjGp, 
vVyVzWKKaCvGClCSagOb>>>( (float*)outputs[0],  (float*)outputs[0], 
hTbrmZSPUORqMIprVbKk, YDoginwuwFxabuYCVqpT, ZqQxEyCjEixByRZYMkbj, 
boZxhUbgnNUlJQXTuWwi, voqEJSkAwmNPuqzoiuom); } else if (hpOzCTZasBMYKoXVxMDZ 
== 1 && hqbKXLMjsDxRQqyJEgbg == 1 && hqVFaqkobRNLQNgtbaai > 1) { 
offset_vector_kernel<<<OuTwywxKeMgznElXdjGp, vVyVzWKKaCvGClCSagOb>>>( 
(float*)outputs[0],  (float*)outputs[0], hTbrmZSPUORqMIprVbKk, 
YmfPcXPXNFZDznkzKZrl, YMNbgnUYZspjMLjwcIOS, 
YDoginwuwFxabuYCVqpT, ZqQxEyCjEixByRZYMkbj, boZxhUbgnNUlJQXTuWwi, 
voqEJSkAwmNPuqzoiuom); } else if (YMNbgnUYZspjMLjwcIOS == 
hqVFaqkobRNLQNgtbaai) { offset_tensor3d_kernel<<<OuTwywxKeMgznElXdjGp, 
vVyVzWKKaCvGClCSagOb>>>( (float*)outputs[0],  (float*)outputs[0], 
hTbrmZSPUORqMIprVbKk, WbTBQxsNsCURmwRhNTAD, WawamKKnqecNqBXIyHIl, 
YmfPcXPXNFZDznkzKZrl, YMNbgnUYZspjMLjwcIOS, 
YDoginwuwFxabuYCVqpT, ZqQxEyCjEixByRZYMkbj, boZxhUbgnNUlJQXTuWwi, 
voqEJSkAwmNPuqzoiuom); } else { 
offset_matrix2d_kernel<<<OuTwywxKeMgznElXdjGp, 
vVyVzWKKaCvGClCSagOb>>>( (float*)outputs[0],  (float*)outputs[0], 
hTbrmZSPUORqMIprVbKk, WbTBQxsNsCURmwRhNTAD, YmfPcXPXNFZDznkzKZrl, 
YMNbgnUYZspjMLjwcIOS, YDoginwuwFxabuYCVqpT, ZqQxEyCjEixByRZYMkbj, 
boZxhUbgnNUlJQXTuWwi, voqEJSkAwmNPuqzoiuom); } return 0; } void 
MWElementwiseAffineLayerImpl::loadScale(const char* sCDdEyIOjXBVHhcakBhd) { 
FILE* SmibqCQPbtzycGEpwhpN = MWCNNLayer::openBinaryFile(sCDdEyIOjXBVHhcakBhd); 
assert(SmibqCQPbtzycGEpwhpN); long int eWYFXrUazhqiEIscccda = 
rZyMIPooLjRiXLgSWDuw*rfIckzovFcUSejaEKNlY*qquNiJHQtfSLDMNCPIBJ; rwPhFWHcKnJsClVtebGW 
= MALLOC_CALL(sizeof(float)*eWYFXrUazhqiEIscccda); call_fread(rwPhFWHcKnJsClVtebGW, 
sizeof(float), eWYFXrUazhqiEIscccda, SmibqCQPbtzycGEpwhpN, sCDdEyIOjXBVHhcakBhd); 
CUDA_CALL(hipMemcpy(pxmnUEWGnfCxJNuDkXAo, rwPhFWHcKnJsClVtebGW, 
sizeof(float)*eWYFXrUazhqiEIscccda, hipMemcpyHostToDevice)); fclose(SmibqCQPbtzycGEpwhpN);  
} void MWElementwiseAffineLayerImpl::loadOffset(const char* 
jLmklYtHcmTxayQTpmRw) { FILE* SmibqCQPbtzycGEpwhpN = 
MWCNNLayer::openBinaryFile(jLmklYtHcmTxayQTpmRw); assert(SmibqCQPbtzycGEpwhpN); long 
int eWYFXrUazhqiEIscccda = 
hpOzCTZasBMYKoXVxMDZ*hqbKXLMjsDxRQqyJEgbg*hjXpmIaeqlKKSDDBSxtE; 
ikTyjLTPRBkBRlLSyxXG = MALLOC_CALL(sizeof(float)*eWYFXrUazhqiEIscccda); 
call_fread(ikTyjLTPRBkBRlLSyxXG, sizeof(float), eWYFXrUazhqiEIscccda, SmibqCQPbtzycGEpwhpN, 
jLmklYtHcmTxayQTpmRw); CUDA_CALL(hipMemcpy(hTbrmZSPUORqMIprVbKk, 
ikTyjLTPRBkBRlLSyxXG, sizeof(float)*eWYFXrUazhqiEIscccda, hipMemcpyHostToDevice)); 
fclose(SmibqCQPbtzycGEpwhpN);  } void MWElementwiseAffineLayerImpl::cleanup() { if 
(pxmnUEWGnfCxJNuDkXAo) { CUDA_FREE_CALL(pxmnUEWGnfCxJNuDkXAo); } if (hTbrmZSPUORqMIprVbKk) 
{ CUDA_FREE_CALL(hTbrmZSPUORqMIprVbKk); } if (rwPhFWHcKnJsClVtebGW) 
free(rwPhFWHcKnJsClVtebGW); if (ikTyjLTPRBkBRlLSyxXG) 
free(ikTyjLTPRBkBRlLSyxXG); }