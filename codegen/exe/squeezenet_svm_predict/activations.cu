#include "hip/hip_runtime.h"
//
// File: activations.cu
//
// GPU Coder version                    : 1.5
// CUDA/C/C++ source code generated on  : 06-Aug-2020 05:06:39
//

// Include Files
#include "activations.h"
#include "DeepLearningNetwork.h"
#include "MWCudaDimUtility.hpp"
#include "rt_nonfinite.h"
#include "squeezenet_svm_predict.h"

// Function Declarations
static __global__ void b_DeepLearningNetwork_activatio(const unsigned char in
  [154587], unsigned char b_in[154587]);
static __global__ void c_DeepLearningNetwork_activatio(const unsigned char in
  [154587], float miniBatchT[154587]);
static __global__ void d_DeepLearningNetwork_activatio(const float outMiniBatch
  [1000], float outT[1000]);

// Function Definitions

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char in[154587]
//                unsigned char b_in[154587]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void b_DeepLearningNetwork_activatio
  (const unsigned char in[154587], unsigned char b_in[154587])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 154587) {
    b_in[i] = in[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char in[154587]
//                float miniBatchT[154587]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void c_DeepLearningNetwork_activatio
  (const unsigned char in[154587], float miniBatchT[154587])
{
  unsigned int threadId;
  int p;
  int i1;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId % 227U);
  threadId = (threadId - static_cast<unsigned int>(i)) / 227U;
  i1 = static_cast<int>(threadId % 227U);
  threadId = (threadId - static_cast<unsigned int>(i1)) / 227U;
  p = static_cast<int>(threadId);
  if (p < 3) {
    miniBatchT[(i + 227 * i1) + 51529 * p] = static_cast<float>(in[(i1 + 227 * i)
      + 51529 * p]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float outMiniBatch[1000]
//                float outT[1000]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void d_DeepLearningNetwork_activatio
  (const float outMiniBatch[1000], float outT[1000])
{
  unsigned int threadId;
  int p;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  p = static_cast<int>(threadId);
  if (p < 1000) {
    outT[p] = outMiniBatch[p];
  }
}

//
// Arguments    : b_squeezenet_0 *obj
//                const unsigned char in[154587]
//                float outT[1000]
// Return Type  : void
//
void DeepLearningNetwork_activations(b_squeezenet_0 *obj, const unsigned char
  in[154587], float outT[1000])
{
  float (*gpu_miniBatchT)[154587];
  float (*gpu_outMiniBatch)[1000];
  unsigned char (*gpu_in)[154587];
  unsigned char (*b_gpu_in)[154587];
  float (*gpu_outT)[1000];
  hipMalloc(&gpu_outT, 4000UL);
  hipMalloc(&gpu_outMiniBatch, 4000UL);
  hipMalloc(&gpu_miniBatchT, 618348UL);
  hipMalloc(&b_gpu_in, 154587UL);
  hipMalloc(&gpu_in, 154587UL);
  hipMemcpy(gpu_in, (void *)&in[0], 154587UL, hipMemcpyHostToDevice);
  b_DeepLearningNetwork_activatio<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_in, *b_gpu_in);
  c_DeepLearningNetwork_activatio<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*b_gpu_in, *gpu_miniBatchT);
  hipMemcpy(obj->getInputDataPointer(), *gpu_miniBatchT, 154587U * sizeof(float),
             hipMemcpyDeviceToDevice);
  obj->activations(66);
  hipMemcpy(*gpu_outMiniBatch, obj->getLayerOutput(66, 0), 1000U * sizeof(float),
             hipMemcpyDeviceToDevice);
  d_DeepLearningNetwork_activatio<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_outMiniBatch, *gpu_outT);
  hipMemcpy(&outT[0], gpu_outT, 4000UL, hipMemcpyDeviceToHost);
  hipFree(*gpu_in);
  hipFree(*b_gpu_in);
  hipFree(*gpu_miniBatchT);
  hipFree(*gpu_outMiniBatch);
  hipFree(*gpu_outT);
}

//
// File trailer for activations.cu
//
// [EOF]
//
