#include "hip/hip_runtime.h"
//
// File: squeezenet_svm_predict.cu
//
// GPU Coder version                    : 1.5
// CUDA/C/C++ source code generated on  : 06-Aug-2020 05:06:39
//

// Include Files
#include "squeezenet_svm_predict.h"
#include "CompactClassificationECOC.h"
#include "DeepLearningNetwork.h"
#include "MWCudaDimUtility.hpp"
#include "activations.h"
#include "hip/hip_math_constants.h"
#include "rt_nonfinite.h"
#include "squeezenet_svm_predict_data.h"
#include "squeezenet_svm_predict_initialize.h"
#include <cstring>

// Type Definitions
struct c_classreg_learning_coder_class
{
  double ClassNames[7];
  int ClassNamesLength[7];
  double Prior[7];
  double Cost[49];
  double CodingMatrix[147];
  char ScoreType[3];
};

struct struct_T
{
  c_classreg_learning_coder_class classifier;
};

// Variable Definitions
static b_squeezenet_0 net;
static bool net_not_empty;
static struct_T v;
__device__ struct_T gpu_v;

// Function Declarations
static __device__ double rt_powd_snf(double u0, double u1);
static __device__ double rt_roundd_snf(double u);
static __global__ void squeezenet_svm_predict_kernel1(const double dv[7]);
static __global__ void squeezenet_svm_predict_kernel10(const double rowWeights
  [2724], const int xoffset, double rowWeightsTotal[227]);
static __global__ void squeezenet_svm_predict_kernel11(const double colWeights
  [2043], double colWeightsTotal[227]);
static __global__ void squeezenet_svm_predict_kernel12(const double colWeights
  [2043], const int xoffset, double colWeightsTotal[227]);
static __global__ void squeezenet_svm_predict_kernel13(const double
  rowWeightsTotal[227], const double rowWeights[2724], const unsigned char in
  [921600], const short ipRowIndices[2724], unsigned char partialResize[326880]);
static __global__ void squeezenet_svm_predict_kernel14(const double
  colWeightsTotal[227], const double colWeights[2043], const unsigned char
  partialResize[326880], const short ipColIndices[2043], unsigned char out
  [154587]);
static __global__ void squeezenet_svm_predict_kernel2(const signed char iv[49]);
static __global__ void squeezenet_svm_predict_kernel3(const signed char iv1[147]);
static __global__ void squeezenet_svm_predict_kernel4(const char cv[3]);
static __global__ void squeezenet_svm_predict_kernel5(short aux1[1280]);
static __global__ void squeezenet_svm_predict_kernel6(short aux2[960]);
static __global__ void squeezenet_svm_predict_kernel7(const short aux1[1280],
  double rowWeights[2724], short ipRowIndices[2724]);
static __global__ void squeezenet_svm_predict_kernel8(const short aux2[960],
  double colWeights[2043], short ipColIndices[2043]);
static __global__ void squeezenet_svm_predict_kernel9(const double rowWeights
  [2724], double rowWeightsTotal[227]);

// Function Definitions

//
// Arguments    : double u0
//                double u1
// Return Type  : double
//
static __device__ double rt_powd_snf(double u0, double u1)
{
  double b_y;
  if ((static_cast<int>(isnan(u0))) || (static_cast<int>(isnan(u1)))) {
    b_y = HIP_NAN;
  } else {
    double d;
    double d1;
    d = fabs(u0);
    d1 = fabs(u1);
    if (isinf(u1)) {
      if (d == 1.0) {
        b_y = 1.0;
      } else if (d > 1.0) {
        if (u1 > 0.0) {
          b_y = HIP_INF;
        } else {
          b_y = 0.0;
        }
      } else if (u1 > 0.0) {
        b_y = 0.0;
      } else {
        b_y = HIP_INF;
      }
    } else if (d1 == 0.0) {
      b_y = 1.0;
    } else if (d1 == 1.0) {
      if (u1 > 0.0) {
        b_y = u0;
      } else {
        b_y = 1.0 / u0;
      }
    } else if (u1 == 2.0) {
      b_y = u0 * u0;
    } else if ((static_cast<int>(u1 == 0.5)) && (static_cast<int>(u0 >= 0.0))) {
      b_y = sqrt(u0);
    } else if ((static_cast<int>(u0 < 0.0)) && (static_cast<int>(u1 > floor(u1))))
    {
      b_y = HIP_NAN;
    } else {
      b_y = pow(u0, u1);
    }
  }

  return b_y;
}

//
// Arguments    : double u
// Return Type  : double
//
static __device__ double rt_roundd_snf(double u)
{
  double b_y;
  if (fabs(u) < 4.503599627370496E+15) {
    if (u >= 0.5) {
      b_y = floor(u + 0.5);
    } else if (u > -0.5) {
      b_y = u * 0.0;
    } else {
      b_y = ceil(u - 0.5);
    }
  } else {
    b_y = u;
  }

  return b_y;
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double dv[7]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void squeezenet_svm_predict_kernel1(
  const double dv[7])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 7) {
    gpu_v.classifier.ClassNames[oldIdx] = static_cast<double>(oldIdx) + 1.0;
    gpu_v.classifier.ClassNamesLength[oldIdx] = 1;
    gpu_v.classifier.Prior[oldIdx] = dv[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeights[2724]
//                const int xoffset
//                double rowWeightsTotal[227]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void squeezenet_svm_predict_kernel10
  (const double rowWeights[2724], const int xoffset, double rowWeightsTotal[227])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 227) {
    rowWeightsTotal[oldIdx] += rowWeights[xoffset + oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeights[2043]
//                double colWeightsTotal[227]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void squeezenet_svm_predict_kernel11
  (const double colWeights[2043], double colWeightsTotal[227])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 227) {
    colWeightsTotal[oldIdx] = colWeights[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeights[2043]
//                const int xoffset
//                double colWeightsTotal[227]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void squeezenet_svm_predict_kernel12
  (const double colWeights[2043], const int xoffset, double colWeightsTotal[227])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 227) {
    colWeightsTotal[oldIdx] += colWeights[xoffset + oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeightsTotal[227]
//                const double rowWeights[2724]
//                const unsigned char in[921600]
//                const short ipRowIndices[2724]
//                unsigned char partialResize[326880]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void squeezenet_svm_predict_kernel13
  (const double rowWeightsTotal[227], const double rowWeights[2724], const
   unsigned char in[921600], const short ipRowIndices[2724], unsigned char
   partialResize[326880])
{
  unsigned int threadId;
  int colIdx;
  int rowIdx;
  double sumVal;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId % 3U);
  threadId = (threadId - static_cast<unsigned int>(oldIdx)) / 3U;
  rowIdx = static_cast<int>(threadId % 227U);
  threadId = (threadId - static_cast<unsigned int>(rowIdx)) / 227U;
  colIdx = static_cast<int>(threadId);
  if (colIdx < 480) {
    unsigned char u;
    sumVal = 0.0;
    for (int l = 0; l < 12; l++) {
      sumVal += static_cast<double>(in[((static_cast<int>(ipRowIndices[rowIdx +
        227 * l]) + 640 * colIdx) + 307200 * oldIdx) - 1]) * (rowWeights[rowIdx
        + 227 * l] / rowWeightsTotal[rowIdx]);
    }

    sumVal = rt_roundd_snf(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }

    partialResize[(rowIdx + 227 * colIdx) + 108960 * oldIdx] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeightsTotal[227]
//                const double colWeights[2043]
//                const unsigned char partialResize[326880]
//                const short ipColIndices[2043]
//                unsigned char out[154587]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void squeezenet_svm_predict_kernel14
  (const double colWeightsTotal[227], const double colWeights[2043], const
   unsigned char partialResize[326880], const short ipColIndices[2043], unsigned
   char out[154587])
{
  unsigned int threadId;
  int colIdx;
  int rowIdx;
  double sumVal;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId % 3U);
  threadId = (threadId - static_cast<unsigned int>(oldIdx)) / 3U;
  rowIdx = static_cast<int>(threadId % 227U);
  threadId = (threadId - static_cast<unsigned int>(rowIdx)) / 227U;
  colIdx = static_cast<int>(threadId);
  if (colIdx < 227) {
    unsigned char u;
    sumVal = 0.0;
    for (int l = 0; l < 9; l++) {
      sumVal += static_cast<double>(partialResize[(rowIdx + 227 * (static_cast<
        int>(ipColIndices[colIdx + 227 * l]) - 1)) + 108960 * oldIdx]) *
        (colWeights[colIdx + 227 * l] / colWeightsTotal[colIdx]);
    }

    sumVal = rt_roundd_snf(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }

    out[(rowIdx + 227 * colIdx) + 51529 * oldIdx] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char iv[49]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void squeezenet_svm_predict_kernel2(
  const signed char iv[49])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 49) {
    gpu_v.classifier.Cost[oldIdx] = static_cast<double>(iv[oldIdx]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char iv1[147]
// Return Type  : void
//
static __global__ __launch_bounds__(160, 1) void squeezenet_svm_predict_kernel3(
  const signed char iv1[147])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 147) {
    gpu_v.classifier.CodingMatrix[oldIdx] = static_cast<double>(iv1[oldIdx]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const char cv[3]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void squeezenet_svm_predict_kernel4(
  const char cv[3])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 3) {
    gpu_v.classifier.ScoreType[oldIdx] = cv[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                short aux1[1280]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void squeezenet_svm_predict_kernel5
  (short aux1[1280])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 1280) {
    if (oldIdx + 1 <= 640) {
      aux1[oldIdx] = static_cast<short>(oldIdx + 1);
    } else {
      aux1[oldIdx] = static_cast<short>(1280 - oldIdx);
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                short aux2[960]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void squeezenet_svm_predict_kernel6
  (short aux2[960])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 960) {
    if (oldIdx + 1 <= 480) {
      aux2[oldIdx] = static_cast<short>(oldIdx + 1);
    } else {
      aux2[oldIdx] = static_cast<short>(960 - oldIdx);
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const short aux1[1280]
//                double rowWeights[2724]
//                short ipRowIndices[2724]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void squeezenet_svm_predict_kernel7(
  const short aux1[1280], double rowWeights[2724], short ipRowIndices[2724])
{
  unsigned int threadId;
  int l;
  int rowIdx;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId % 12U);
  rowIdx = static_cast<int>((threadId - static_cast<unsigned int>(k)) / 12U);
  if (rowIdx < 227) {
    double absx2;
    double absx;
    int oldIdx;
    double sumVal;
    sumVal = (static_cast<double>(rowIdx) + 1.0) / 0.3546875 +
      -0.90969162995594721;
    oldIdx = static_cast<int>(floor(sumVal - 5.6387665198237888));
    absx = fabs(0.3546875 * (sumVal - (static_cast<double>(oldIdx + k) + 1.0)));
    absx2 = absx * absx;
    sumVal = rt_powd_snf(absx, 3.0);
    oldIdx = (oldIdx + k) + 1;
    if (oldIdx - 1 == 0) {
      l = 0;
    } else {
      l = static_cast<int>(fmod(static_cast<double>(oldIdx) - 1.0, 1280.0));
      if ((static_cast<int>(l != 0)) && (static_cast<int>(oldIdx - 1 < 0))) {
        l += 1280;
      }
    }

    ipRowIndices[rowIdx + 227 * k] = aux1[l];
    rowWeights[rowIdx + 227 * k] = 0.3546875 * (((1.5 * sumVal - 2.5 * absx2) +
      1.0) * static_cast<double>(absx <= 1.0) + (((-0.5 * sumVal + 2.5 * absx2)
      - 4.0 * absx) + 2.0) * static_cast<double>((static_cast<int>(1.0 < absx)) &&
      (static_cast<int>(absx <= 2.0))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const short aux2[960]
//                double colWeights[2043]
//                short ipColIndices[2043]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void squeezenet_svm_predict_kernel8(
  const short aux2[960], double colWeights[2043], short ipColIndices[2043])
{
  unsigned int threadId;
  int l;
  int colIdx;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId % 9U);
  colIdx = static_cast<int>((threadId - static_cast<unsigned int>(k)) / 9U);
  if (colIdx < 227) {
    double absx2;
    double absx;
    int oldIdx;
    double sumVal;
    sumVal = (static_cast<double>(colIdx) + 1.0) / 0.47291666666666665 +
      -0.55726872246696035;
    oldIdx = static_cast<int>(floor(sumVal - 4.2290748898678414));
    absx = fabs(0.47291666666666665 * (sumVal - (static_cast<double>(oldIdx + k)
      + 1.0)));
    absx2 = absx * absx;
    sumVal = rt_powd_snf(absx, 3.0);
    oldIdx = (oldIdx + k) + 1;
    if (oldIdx - 1 == 0) {
      l = 0;
    } else {
      l = static_cast<int>(fmod(static_cast<double>(oldIdx) - 1.0, 960.0));
      if ((static_cast<int>(l != 0)) && (static_cast<int>(oldIdx - 1 < 0))) {
        l += 960;
      }
    }

    ipColIndices[colIdx + 227 * k] = aux2[l];
    colWeights[colIdx + 227 * k] = 0.47291666666666665 * (((1.5 * sumVal - 2.5 *
      absx2) + 1.0) * static_cast<double>(absx <= 1.0) + (((-0.5 * sumVal + 2.5 *
      absx2) - 4.0 * absx) + 2.0) * static_cast<double>((static_cast<int>(1.0 <
      absx)) && (static_cast<int>(absx <= 2.0))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeights[2724]
//                double rowWeightsTotal[227]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void squeezenet_svm_predict_kernel9(
  const double rowWeights[2724], double rowWeightsTotal[227])
{
  unsigned int threadId;
  int oldIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 227) {
    rowWeightsTotal[oldIdx] = rowWeights[oldIdx];
  }
}

//
// Arguments    : const unsigned char in[921600]
// Return Type  : double
//
double squeezenet_svm_predict(const unsigned char in[921600])
{
  double out;
  static const signed char iv[49] = { 0, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1,
    1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1,
    0, 1, 1, 1, 1, 1, 1, 1, 0 };

  static const double dv[7] = { 0.019064124783362217, 0.090121317157712308,
    0.20537261698440207, 0.15251299826689774, 0.20103986135181975,
    0.23743500866551126, 0.094454072790294621 };

  static const signed char iv1[147] = { 1, -1, 0, 0, 0, 0, 0, 1, 0, -1, 0, 0, 0,
    0, 1, 0, 0, -1, 0, 0, 0, 1, 0, 0, 0, -1, 0, 0, 1, 0, 0, 0, 0, -1, 0, 1, 0, 0,
    0, 0, 0, -1, 0, 1, -1, 0, 0, 0, 0, 0, 1, 0, -1, 0, 0, 0, 0, 1, 0, 0, -1, 0,
    0, 0, 1, 0, 0, 0, -1, 0, 0, 1, 0, 0, 0, 0, -1, 0, 0, 1, -1, 0, 0, 0, 0, 0, 1,
    0, -1, 0, 0, 0, 0, 1, 0, 0, -1, 0, 0, 0, 1, 0, 0, 0, -1, 0, 0, 0, 1, -1, 0,
    0, 0, 0, 0, 1, 0, -1, 0, 0, 0, 0, 1, 0, 0, -1, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0,
    0, 1, 0, -1, 0, 0, 0, 0, 0, 1, -1 };

  static const char cv[3] = { 'i', 'n', 'f' };

  int k;
  static unsigned char b_out[154587];
  float featuresTest[1000];
  float score[7];
  double (*gpu_dv)[7];
  signed char (*gpu_iv)[49];
  signed char (*gpu_iv1)[147];
  char (*gpu_cv)[3];
  short (*gpu_aux1)[1280];
  short (*gpu_aux2)[960];
  double (*gpu_rowWeights)[2724];
  short (*gpu_ipRowIndices)[2724];
  double (*gpu_colWeights)[2043];
  short (*gpu_ipColIndices)[2043];
  double (*gpu_rowWeightsTotal)[227];
  double (*gpu_colWeightsTotal)[227];
  unsigned char (*gpu_in)[921600];
  unsigned char (*gpu_partialResize)[326880];
  unsigned char (*gpu_out)[154587];
  double b[7];
  double c[7];
  double d[147];
  if (!isInitialized_squeezenet_svm_predict) {
    squeezenet_svm_predict_initialize();
  }

  hipMalloc(&gpu_out, 154587UL);
  hipMalloc(&gpu_partialResize, 326880UL);
  hipMalloc(&gpu_ipColIndices, 4086UL);
  hipMalloc(&gpu_in, 921600UL);
  hipMalloc(&gpu_ipRowIndices, 5448UL);
  hipMalloc(&gpu_colWeightsTotal, 1816UL);
  hipMalloc(&gpu_colWeights, 16344UL);
  hipMalloc(&gpu_rowWeightsTotal, 1816UL);
  hipMalloc(&gpu_rowWeights, 21792UL);
  hipMalloc(&gpu_aux2, 1920UL);
  hipMalloc(&gpu_aux1, 2560UL);
  hipMalloc(&gpu_cv, 3UL);
  hipMalloc(&gpu_iv1, 147UL);
  hipMalloc(&gpu_iv, 49UL);
  hipMalloc(&gpu_dv, 56UL);

  // keep this function definition separate and not merge with the calling function  
  if (!net_not_empty) {
    DeepLearningNetwork_setup(&net);
    net_not_empty = true;
    hipMemcpy(gpu_dv, (void *)&dv[0], 56UL, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_v), &v, 1720UL, 0UL, hipMemcpyHostToDevice);
    squeezenet_svm_predict_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_dv);
    hipMemcpyFromSymbol(&v, HIP_SYMBOL(gpu_v), 1720UL, 0UL, hipMemcpyDeviceToHost);
    hipMemcpy(gpu_iv, (void *)&iv[0], 49UL, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_v), &v, 1720UL, 0UL, hipMemcpyHostToDevice);
    squeezenet_svm_predict_kernel2<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
      (*gpu_iv);
    hipMemcpyFromSymbol(&v, HIP_SYMBOL(gpu_v), 1720UL, 0UL, hipMemcpyDeviceToHost);
    hipMemcpy(gpu_iv1, (void *)&iv1[0], 147UL, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_v), &v, 1720UL, 0UL, hipMemcpyHostToDevice);
    squeezenet_svm_predict_kernel3<<<dim3(1U, 1U, 1U), dim3(160U, 1U, 1U)>>>
      (*gpu_iv1);
    hipMemcpyFromSymbol(&v, HIP_SYMBOL(gpu_v), 1720UL, 0UL, hipMemcpyDeviceToHost);
    hipMemcpy(gpu_cv, (void *)&cv[0], 3UL, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_v), &v, 1720UL, 0UL, hipMemcpyHostToDevice);
    squeezenet_svm_predict_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_cv);
    hipMemcpyFromSymbol(&v, HIP_SYMBOL(gpu_v), 1720UL, 0UL, hipMemcpyDeviceToHost);
  }

  squeezenet_svm_predict_kernel5<<<dim3(3U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_aux1);
  squeezenet_svm_predict_kernel6<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_aux2);
  squeezenet_svm_predict_kernel7<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_aux1, *gpu_rowWeights, *gpu_ipRowIndices);
  squeezenet_svm_predict_kernel8<<<dim3(4U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_aux2, *gpu_colWeights, *gpu_ipColIndices);
  squeezenet_svm_predict_kernel9<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
    (*gpu_rowWeights, *gpu_rowWeightsTotal);
  for (k = 0; k < 11; k++) {
    squeezenet_svm_predict_kernel10<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
      (*gpu_rowWeights, (k + 1) * 227, *gpu_rowWeightsTotal);
  }

  squeezenet_svm_predict_kernel11<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
    (*gpu_colWeights, *gpu_colWeightsTotal);
  for (k = 0; k < 8; k++) {
    squeezenet_svm_predict_kernel12<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
      (*gpu_colWeights, (k + 1) * 227, *gpu_colWeightsTotal);
  }

  hipMemcpy(gpu_in, (void *)&in[0], 921600UL, hipMemcpyHostToDevice);
  squeezenet_svm_predict_kernel13<<<dim3(639U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_rowWeightsTotal, *gpu_rowWeights, *gpu_in, *gpu_ipRowIndices,
     *gpu_partialResize);
  squeezenet_svm_predict_kernel14<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_colWeightsTotal, *gpu_colWeights, *gpu_partialResize,
     *gpu_ipColIndices, *gpu_out);
  hipMemcpy(&b_out[0], gpu_out, 154587UL, hipMemcpyDeviceToHost);
  DeepLearningNetwork_activations(&net, b_out, featuresTest);
  for (int e = 0; e < 7; e++) {
    b[e] = v.classifier.ClassNames[e];
  }

  for (int f = 0; f < 7; f++) {
    c[f] = v.classifier.Prior[f];
  }

  std::memcpy(&d[0], &v.classifier.CodingMatrix[0], 147U * sizeof(double));
  c_CompactClassificationECOC_pre(b, c, d, featuresTest, &out, score);

  //  score(out)
  hipFree(*gpu_dv);
  hipFree(*gpu_iv);
  hipFree(*gpu_iv1);
  hipFree(*gpu_cv);
  hipFree(*gpu_aux1);
  hipFree(*gpu_aux2);
  hipFree(*gpu_rowWeights);
  hipFree(*gpu_rowWeightsTotal);
  hipFree(*gpu_colWeights);
  hipFree(*gpu_colWeightsTotal);
  hipFree(*gpu_ipRowIndices);
  hipFree(*gpu_in);
  hipFree(*gpu_ipColIndices);
  hipFree(*gpu_partialResize);
  hipFree(*gpu_out);
  return out;
}

//
// Arguments    : void
// Return Type  : void
//
void squeezenet_svm_predict_init()
{
  net_not_empty = false;
}

//
// File trailer for squeezenet_svm_predict.cu
//
// [EOF]
//
