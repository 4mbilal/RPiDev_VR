//
// File: squeezenet_svm_predict_initialize.cu
//
// GPU Coder version                    : 1.5
// CUDA/C/C++ source code generated on  : 06-Aug-2020 05:06:39
//

// Include Files
#include "squeezenet_svm_predict_initialize.h"
#include "DeepLearningNetwork.h"
#include "rt_nonfinite.h"
#include "squeezenet_svm_predict.h"
#include "squeezenet_svm_predict_data.h"

// Function Declarations
static void cublasEnsureInitialization();

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//
static void cublasEnsureInitialization()
{
  if (cublasGlobalHandle == NULL) {
    hipblasCreate(&cublasGlobalHandle);
    hipblasSetPointerMode(cublasGlobalHandle, HIPBLAS_POINTER_MODE_DEVICE);
  }
}

//
// Arguments    : void
// Return Type  : void
//
void squeezenet_svm_predict_initialize()
{
  rt_InitInfAndNaN();
  squeezenet_svm_predict_init();
  cublasEnsureInitialization();
  isInitialized_squeezenet_svm_predict = true;
}

//
// File trailer for squeezenet_svm_predict_initialize.cu
//
// [EOF]
//
